#include "hip/hip_runtime.h"
/*
Copyright 2008 Steven Brodhead, Jr.
Copyright 2011-2016 Steven Brodhead, Sr., Centcom Inc.

// All rights reserved.
 
//     Fractal Architect Render Engine - a GPU accelerated flame fractal renderer written in C++
//
//     This is free software; you can redistribute it and/or modify it under the terms of the GNU Lesser
//     General Public License as published by the Free Software Foundation; either version 2.1 of the
//     License, or (at your option) any later version.
//
//     This software is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without
//     even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
//     Lesser General Public License for more details.
//
//     You should have received a copy of the GNU Lesser General Public License along with this software;
//     if not, write to the Free Software Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA
//     02110-1301 USA, or see the FSF site: http://www.fsf.org.
*/
#define NUM_ITERATIONS 100
// #define DENSITY_KERNAL_RADIUS 7
#define DENSITY_KERNAL_RADIUS_16KB 7
#define DENSITY_KERNAL_RADIUS_32KB 14
#define DENSITY_KERNAL_RADIUS_48KB 19
#define NUM_FRAMES 160
#define FRAME_RATE 30
#define BITRATE 54000000

#ifndef SUPERSAMPLE_WIDTH
#define SUPERSAMPLE_WIDTH 0.25f
#endif

#ifndef FLAMEDATA_H
#define FLAMEDATA_H

#define MAX_XFORMS 58 // We're limited to 64KB constant memory for compute capacity 1.0.
// All xForms must fit in this.


#define NO_RGBA_CONSTRUCTOR

#define uint  unsigned
#define ulong unsigned long

#define M_PI_F    3.141592653589793f
#define M_PI_2_F  1.5707963267949f
#define M_PI_4_F  0.78539816339745f
#define M_1_PI_F  0.31830988618379f
#define M_2_PI_F  0.63661977236758f

#define JWF_EXTENSIONS

#ifndef RGBA_H
#define RGBA_H

struct  __align__(16) rgba
{
    float r;
    float g;
    float b;
    float a;
};

#endif

#ifdef JWF_EXTENSIONS
__device__ float lerpf(float a, float b, float p) {
    return a + (b - a) * p;
}

__device__ float sqrtf_safe(float x) {
  if (x <= 0.0f)
    return 0.0f;
  else	
    return sqrtf(x);
}


struct __align__(8) Complex
{
  float per_fix;
  float re;
  float im;
  float save_re;
  float save_im;
};

__device__ void Complex_Init(Complex *c, float Rp, float Ip) {
  c->re = Rp;
  c->im = Ip;
  c->save_re = 0.f;
  c->save_im = 0.f;
  c->per_fix = 0.f;  
}

__device__ float Complex_Mag2(Complex *c) {
    return c->re * c->re + c->im * c->im;
}
  
__device__ float Complex_MagInv(Complex *c) {
    float M2 = Complex_Mag2(c);
    return (M2 < 1e-10 ? 1.0f : 1.0f / M2);
}
  
__device__ void Complex_Recip(Complex *c) {
    float mi = Complex_MagInv(c);
    c->re = c->re * mi;
    c->im = -c->im * mi;
}

__device__ void Complex_Dec(Complex *c) {
  c->re -= 1.0f;
}

__device__ void Complex_Inc(Complex *c) {
  c->re += 1.0f;
}

__device__ void Complex_Neg(Complex *c) {
  c->re = -c->re;
  c->im = -c->im;
}
  
__device__ void Complex_Div(Complex *c, Complex *zz) {
  float r2 = c->im * zz->im + c->re * zz->re;
  float i2 = c->im * zz->re - c->re * zz->im;
  float M2 = Complex_MagInv(zz);
  c->re = r2 * M2;
  c->im = i2 * M2;
}
  
__device__ void Complex_Copy(Complex *c, Complex *zz) {
  c->re = zz->re;
  c->im = zz->im;
}
  
__device__ float Complex_Mag2eps(Complex *c) {
    return c->re * c->re + c->im * c->im + 1e-10;
}

__device__ float Complex_Arg(Complex *c) {
  return (c->per_fix + atan2f(c->im, c->re));
}

__device__ void Complex_Log(Complex *c) {
  Complex L_eps;
  Complex_Init(&L_eps, 0.5f * logf(Complex_Mag2eps(c)), Complex_Arg(c));
  Complex_Copy(c, &L_eps);
}

__device__ void Complex_Scale(Complex *c, float mul) {
    c->re = c->re * mul;
    c->im = c->im * mul;
}
  
__device__ void Complex_AtanH(Complex *c) {
    Complex D;
	Complex_Init(&D, c->re, c->im);
    Complex_Dec(&D);
    Complex_Neg(&D);
    Complex_Inc(c);
    Complex_Div(c, &D);
    Complex_Log(c);
    Complex_Scale(c, 0.5f);
}

__device__ void Complex_AcotH(Complex *c) {
   Complex_Recip(c);
   Complex_AtanH(c);
}

__device__ void Complex_Flip(Complex *c) {
    float r2 = c->im;
    float i2 = c->re;
    c->re = r2;
    c->im = i2;
  }
  
__device__ void Complex_Sqr(Complex *c) {
  float r2 = c->re * c->re - c->im * c->im;
  float i2 = 2.f * c->re * c->im;
  c->re = r2;
  c->im = i2;
}  

  
__device__ void Complex_Add(Complex *c, Complex *zz) {
  c->re += zz->re;
  c->im += zz->im;
}

__device__ void Complex_Sub(Complex *c, Complex *zz) {
  c->re -= zz->re;
  c->im -= zz->im;
}


__device__ void Complex_Mul(Complex *c, Complex *zz) {
   if (zz->im == 0.0) {
      Complex_Scale(c, zz->re);
      return;
   }
   float  r2 = c->re * zz->re - c->im * zz->im;
   float  i2 = c->re * zz->im + c->im * zz->re;
   c->re = r2;
   c->im = i2;
}
    
  
__device__ void Complex_One(Complex *c) {
  c->re = 1.0f;
  c->im = 0.0f;
}

__device__ void Complex_Conj(Complex *c) {
  c->im = -c->im;
}

__device__ float Complex_Radius(Complex *c) {
    return hypotf(c->re, c->im);
}

__device__ void Complex_Sqrt(Complex *c) {
  float Rad = Complex_Radius(c);
  float sb = (c->im < 0) ? -1.f : 1.f;
  c->im = sb * sqrtf(0.5f * (Rad - c->re));
  c->re = sqrtf(0.5f * (Rad + c->re));
  if (c->per_fix < 0)
    Complex_Neg(c);
}
  
  
__device__ void Complex_ToP(Complex *c, Complex *dst) {
  Complex_Init(dst, Complex_Radius(c), Complex_Arg(c));
}
  
  
__device__ void Complex_UnP(Complex *c, Complex *dst) {
  Complex_Init(dst, c->re * cosf(c->im), c->re * sinf(c->im));
}  
  
__device__ void Complex_Pow(Complex *c, float exp) {
    if (exp == 0.0f) {
      Complex_One(c);
      return;
    }
    float ex = fabsf(exp);
    if (exp < 0) {
      Complex_Recip(c);
    }
    if (ex == 0.5f) {
      Complex_Sqrt(c);
      return;
    }
    if (ex == 1.0f) {
      return;
    }
    if (ex == 2.0f) {
      Complex_Sqr(c);
      return;
    }
    // In general we need sin, cos etc
    Complex PF;
    Complex_ToP(c, &PF);
    PF.re = powf(PF.re, ex);
    PF.im = PF.im * ex;
	
	Complex PFU;	
	Complex_UnP(&PF, &PFU);	
    Complex_Copy(c, &PFU);
  }
  
 
__device__ void Complex_AsinH(Complex *c) {
  Complex D;
  Complex_Init(&D, c->re, c->im);
  Complex_Sqr(&D);
  Complex_Inc(&D);
  Complex_Pow(&D, 0.5f);
  Complex_Add(c, &D);
  Complex_Log(c);
}

__device__ void Complex_AcosH(Complex *c) {
  Complex D;
  Complex_Init(&D, c->re, c->im);
  Complex_Sqr(&D);
  Complex_Dec(&D);
  Complex_Pow(&D, 0.5f);
  Complex_Add(c, &D);
  Complex_Log(c);
}

__device__ void Complex_AcosecH(Complex *c) {
   Complex_Recip(c);
   Complex_AcosH(c);
}

  
#endif

struct __align__(8) xForm
{
    float a;
    float b;
    float c;
    float d;
    float e;
    float f;
    float pa;
    float pb;
    float pc;
    float pd;
    float pe;
    float pf;
    float color;
    float symmetry;
    float weight;
    float opacity;
    float var_color;
    int   rotates;
#ifdef JWF_EXTENSIONS	
    float yzA;
    float yzB;
    float yzC;
    float yzD;
    float yzE;
    float yzF;
    float yzPa;
    float yzPb;
    float yzPc;
    float yzPd;
    float yzPe;
    float yzPf;
    float zxA;
    float zxB;
    float zxC;
    float zxD;
    float zxE;
    float zxF;
    float zxPa;
    float zxPb;
    float zxPc;
    float zxPd;
    float zxPe;
    float zxPf;
	int useXyz;	
#endif	
};

// each xform has a variable length list of active variations and each variation has its own specific variable sized varpar struct
struct __align__(16) VariationListNode
{                        // all of the lists are concatenated into a single buffer - a separate xformUsageIndex has the offset to the xform's first variation in this list
    uint variationID;    // the numeric value identifying the variation from the variation set - NOTE id of zero is used to signify end of list
    uint varparOffset;   // the offset in varpar union list for this variation's specific varpar struct
    uint enterGroup;     // the state transition that handles entering Pre, Normal, and Post variation groups
};

struct __align__(8) unAnimatedxForm
{
    float a;
    float b;
    float d;
    float e;
    int rotates;
};

struct __align__(16) FlameParams
{
    struct rgba background;
    float center[2];                //{x,y}
    float size[2];                    //size/(scale*zoom)
    float scale;
    float zoom;
    float cam_yaw;
    float cam_pitch;
    float cam_perspective;
    int   clipToNDC;
    float cam_dof;
    float cam_zpos;
    float cam_x;
    float cam_y;
    float cam_z;
    float cam_fov;
    float cam_near;
    float cam_orthowide;
    float hue;
    float numBatches;
    float quality;
    float desiredQuality;
    float rotation;
    float symmetryKind;
    float brightness;
    float gamma;
    float gammaThreshold;
    float alphaGamma;
    float vibrancy;
    unsigned int   numTrans;
    unsigned int   numFinal;
    int   useFinal;
    float supersampleWidth;
    int   frame;
    int   useXaos;
    int   oversample;
    float   highlightPower;
    int    estimatorRadius;			// default 7
    float  estimatorCurve;			// default 0.4
#ifdef JWF_EXTENSIONS
	float cam_roll;
	float cam_bank;
	float cam_xfocus;
	float cam_yfocus;
	float cam_zfocus;
	float cam_xpos;
	float cam_ypos;
	float cam_dist;
	float cam_dof_exponent;
	float cam_dof_area;
	bool legacy_dof;
	int dof_type;
	float dof_scale;
	float dof_fade;
#endif
};


struct Flame //  : public std::enable_shared_from_this<Flame>
{
    struct FlameParams params;
    int                numColors;
    struct xForm      *trans;
    struct xForm      *finals;
    struct rgba       *colorIndex;
    float             *colorLocations;
    float             *switchMatrix;
    //  std::vector<SharedVariationChain> xformVarChains;
    //  std::vector<SharedVariationChain> finalVarChains;
//#ifdef __cplusplus
//    Flame();
//    Flame(int numTrans,int paletteSize, int numFinal);
//    Flame(int numTrans,int paletteSize, int numFinal, int alignment);
//    Flame(const Flame &other);
//    Flame(const Flame &other, int alignment);
//    
//    void Clone(Flame** target);
//    void CloneAligned(Flame** target, int alignment);
//    void deleteChildren();
//    void prepareSwitchMatrix (float *brick);
//    ~Flame();
//#endif
} __attribute__ ((aligned (16)));
#endif

__VARPAR_STRUCT_DECLS__


#define PI 3.141592653589793f

#ifndef FLAM4_KERNAL_CUH
#define FLAM4_KERNAL_CUH

struct __align__(16) point
{
    float x;
    float y;
    float z;
    float pal;
#ifdef JWF_EXTENSIONS
	float colorR, colorG, colorB, colorA;
	bool useRgb;
	bool doHide;
#endif
};
#endif

#ifndef FOR_2D
struct  __align__(16) CameraViewProperties
{
    float matrix[16];
    float yaw;
    float pitch;
    float roll;
    float perspective;
    float dof;
    float zpos;
    float cosRoll;
    float sinRoll;
    float camWidth;
    float camHeight;
    float centerX;
    float centerY;
    int   clipToNDC;
    float rotatedViewOffsetx;
    float rotatedViewOffsety;
#ifdef JWF_EXTENSIONS
	float bank;
	float focusX;
	float focusY;
	float focusZ;
	float camPosX;
	float camPosY;
	float camPosZ;
	float camDist;
	float camDOFExponent;
	float camDOFArea;
	float camDOF;
	bool legacyDOF;
	int dofType;
	float dofScale;
	float dofFade;	
#endif	
};
#endif

// so it can be precompiled as part of the build for syntax checking
#ifndef KERNEL_RUNTIME
#define WARP_SIZE 32
#define NUM_POINTS 64
#endif
#define ADD_EPSILON +epsilon
//#define ADD_EPSILON +1.e-7f
//#define ADD_EPSILON +1.e-10f
//#define ADD_EPSILON

#define WARPS_PER_BLOCK 2
#define BLOCKDIM WARPS_PER_BLOCK*WARP_SIZE

#define RANDFLOAT() randFloat(randStates)
#define RANDINT()   randInt(randStates)

__VARIATION_INDEX_DEFINES__

__constant__ FlameParams d_g_Flame;
__constant__ xForm d_g_Xforms[MAX_XFORMS+MAX_XFORMS];  // has both normal & final xforms

__constant__ unsigned int shift1[4] = {6, 2, 13, 3};
__constant__ unsigned int shift2[4] = {13, 27, 21, 12};
__constant__ unsigned int shift3[4] = {18, 2, 7, 13};
__constant__ unsigned int offset[4] = {4294967294, 4294967288, 4294967280, 4294967168};

texture<uchar4, hipTextureType1D, hipReadModeNormalizedFloat> texRef;

__VARIATION_FUNCTION_PROTOTYPES__


__device__ float sqrf(float x) {
  return x*x;
} 

__device__ unsigned int TausStep(unsigned int z, int S1, int S2, int S3, unsigned int M)
{
    unsigned int b = (((z << S1) ^ z) >> S2);
    return (((z &M) << S3) ^ b);
}

__device__ unsigned int randInt(unsigned int *randStates)
{
    unsigned int index = threadIdx.x;
    randStates[index&(WARP_SIZE-1)] = TausStep(randStates[index&(WARP_SIZE-1)], shift1[index&3], shift2[index&3],shift3[index&3],offset[index&3]);
    return (randStates[(index)&(WARP_SIZE-1)]^randStates[(index+1)&(WARP_SIZE-1)]^randStates[(index+2)&(WARP_SIZE-1)]^randStates[(index+3)&(WARP_SIZE-1)]);
}

__device__ float randFloat(unsigned int *randStates)
//This function returns a random float in [0,1] and updates seed
{
    unsigned int y = randInt(randStates);
    return __int_as_float((y&0x007FFFFF)|0x3F800000)-1.0f;
}

__device__ float randFloatWarp(unsigned int *randStates, uint index)
//This function is a workaround for getting a warp wide rand number
{
    randInt(randStates);
    return __int_as_float((randStates[index]&0x007FFFFF)|0x3F800000)-1.0f;
}

__device__ int linear_range_search(float* X, float x, int n)
{
    int index = 256;
    for (int j = n-1; j >= 0; j--) {
        index = (x <= X[j+1]) ? j : index;
    }
    return index;
}

__device__ float curveAdjust(float x,
        float* X,
        float* A,
        float* B,
        float* C,
        float* D,
        uint cpCount)
{
    int index    = linear_range_search(X, x, cpCount - 1);
    float result = (x <= X[0]) ? A[0] : A[cpCount - 1];

    if (index >= 0 && index < (int)cpCount - 1) {
        float t = x - X[index];
        result = A[index] + B[index]*t + C[index]*t*t + D[index]*t*t*t;
    }
    return result;
}


__device__ float4 RGBtoHSV(float4 color)
{
    float r = color.x;
    float g = color.y;
    float b = color.z;
    float mx = fmaxf(fmaxf(r,g),b);
    float mn = fminf(fminf(r,g),b);
    float h,s,v;
    if (mx == mn)
        h = 0.0f;
    else if (mx == r)
        h = .16666666667f*(g-b)/(mx-mn);
    else if (mx == g)
        h = .16666666667f*(b-r)/(mx-mn)+.33333333f;
    else
        h = .16666666667f*(r-g)/(mx-mn)+.66666667f;
    h = h-floorf(h);
    if (mx == 0.0f)
        s = 0.0f;
    else
        s = (mx-mn)/(mx);
    v = mx;
    if (v > 1.0f) // clamp to 1.f if to high value
        v = 1.0f;
    return make_float4(h,s,v,color.w);
}

__device__ float4 RGBtoHSVHueAdjusted(float4 color)
{
    float r = color.x;
    float g = color.y;
    float b = color.z;
    float mx = fmaxf(fmaxf(r,g),b);
    float mn = fminf(fminf(r,g),b);
    float h,s,v;
    if (mx == mn)
        h = 0.0f;
    else if (mx == r)
        h = .16666666667f*(g-b)/(mx-mn);
    else if (mx == g)
        h = .16666666667f*(b-r)/(mx-mn)+.33333333f;
    else
        h = .16666666667f*(r-g)/(mx-mn)+.66666667f;
    h = h-floorf(h);
    if (mx == 0.0f)
        s = 0.0f;
    else
        s = (mx-mn)/(mx);
    v = mx;
    if (v > 1.0f)
    {
        if (h < .33333333f)
        {
            h += (.16666667f-h)*(1.0f-powf(.75f,v-1.0f));
        }
        else if (h < 0.5f)
        {
            h += (h-0.5f)*(1.0f-powf(.75f,v-1.0f));
        }
        else if (h > 0.8333333f)
        {
            h += (h-0.8333333f)*(1.0f-powf(.75f,v-1.0f));
        }
        //float l = .2126f*r+.7152f*g+.0722f*b;
        //float l = (40.0f*r+20.0f*g+b)/61.0f;
        float l = 0.4f+0.4f*cosf(2.0f*PI*(h-0.16666666667f));
        s = fminf(s*powf(1.0f/v,0.6f*(1.0f-l)),s);
    }
    return make_float4(h,s,v,color.w);
}

__device__ float4 HSVtoRGB(float4 color)
{
    float h = color.x;
    float s = color.y;
    float v = color.z;
    float r,g,b;
    int hi = ((int)floorf(h*6.0f))%6;
    float f = h*6.0f-floorf(h*6.0f);
    float p = v*(1.0f-s);
    float q = v*(1.0f-f*s);
    float t = v*(1.0f-(1.0f-f)*s);
    switch (hi)
    {
        case 0:
        {
            r = v;
            g = t;
            b = p;
        }break;
        case 1:
        {
            r = q;
            g = v;
            b = p;
        }break;
        case 2:
        {
            r = p;
            g = v;
            b = t;
        }break;
        case 3:
        {
            r = p;
            g = q;
            b = v;
        }break;
        case 4:
        {
            r = t;
            g = p;
            b = v;
        }break;
        case 5:
        {
            r = v;
            g = p;
            b = q;
        }break;
    }
    return make_float4(r,g,b,color.w);
}


__device__ float4 read_imageStepMode(float4 * image, int length, float index)
{
    float clampedIndex = index - floorf(index);
    float scaledIndex = clampedIndex*(float)(length - 1);
    int iLow = floorf(scaledIndex);
    return image[iLow];
}

__device__ float sinhcoshf(float theta, float* ch)
{
    float expt = expf(theta);
    float exptinv = 1.0f / expt;
    *ch =  (expt + exptinv) * 0.5f;
    return (expt - exptinv) * 0.5f;
}

__VARIATION_FUNCTIONS__


__device__ void iteratePoint(struct VariationListNode *varUsageList,
                float *varpars,
                struct xForm* xform,
                uint xformIndex,
                float epsilon,
                struct point *fromPoint,
                struct point *activePoint,
                unsigned int *randStates,
                uint *permutations,
                float4 *gradients)
{
    unsigned int index = threadIdx.x;
    activePoint[index] = *fromPoint;

    float s0 = xform->symmetry;
    float s1 = .5f-.5f*s0;
    float __pal = (activePoint[index].pal+xform->color)*s1+activePoint[index].pal*s0;
    float pal0 = __pal;

     float __x, __y, __z;
	 float __colorR=0.f, __colorG=0.f, __colorB=0.f, __colorA=0.f;
	 bool __doHide = false;
	 bool __useRgb = false;
	
	 if(xform->useXyz) {
		 __x = xform->a*activePoint[index].x+xform->b*activePoint[index].y;
		 __y = xform->d*activePoint[index].x+xform->e*activePoint[index].y;
		 __z = activePoint[index].z;

         float nx, ny, nz;  
         ny = xform->yzA * __y + xform->yzB * __z;
         nz = xform->yzD * __y + xform->yzE * __z;
         __y = ny;
         __z = nz;

         nx = xform->zxA * __x + xform->zxB * __z;
         nz = xform->zxD * __x + xform->zxE * __z;
         __x = nx;
         __z = nz;
		 
		 __x += xform->c + xform->zxC;
		 __y += xform->f + xform->yzC;
		 __z += xform->yzF + xform->zxF;
	 }
	 else {
		 __x = xform->a*activePoint[index].x+xform->b*activePoint[index].y+xform->c;
		 __y = xform->d*activePoint[index].x+xform->e*activePoint[index].y+xform->f;
		 __z = activePoint[index].z; // 3d hack does not transform them here
	 }

	float __r2, __r, __rinv, __phi, __theta;
    float __px = __x;  // note that enterGroup action will handle resetting these to zero -- also works correctly for xforms with NO variations set
    float __py = __y;
    float __pz = __z;

	float __x0 = __x;
	float __y0 = __y;
	float __z0 = __z;
	
	bool __was_pre = 0;


			__r2 = __x * __x + __y * __y;
			__r = sqrtf(__r2);
			__rinv = 1.0f / __r;

			__phi = atan2f(__x, __y);
			__theta = .5f*PI - __phi;
			if (__theta > PI)
				__theta -= 2.0f*PI;


     __px = 0.f;
     __py = 0.f;
     __pz = 0.f;


    uint varIndex = 0;
    while ((varIndex = varUsageList->variationID) != 0) {
        float *varparCluster = &varpars[varUsageList->varparOffset];
        switch (varIndex) {
            //Now apply the Variations
            __VARIATION_SWITCH_CASES__
            default:
            break;
        }
        varUsageList++;
    }

    if(xform->useXyz) {
      float px = xform->pa*__px+xform->pb*__py;
      float py = xform->pd*__px+xform->pe*__py;
      float pz = __pz;
	  
	  float nx, ny, nz;
	    
      ny = xform->yzPa * py + xform->yzPb * pz;
      nz = xform->yzPd * py + xform->yzPe * pz;
      py = ny;
      pz = nz;
	 
	  nx = xform->zxPa * px + xform->zxPb * pz;
      nz = xform->zxPd * px + xform->zxPe * pz;
      px = nx;
      pz = nz;
	  
      activePoint[index].x = px + xform->pc + xform->zxPc;
      activePoint[index].y = py + xform->pf + xform->yzPc;
      activePoint[index].z = pz + xform->yzPf + xform->zxPf;
	 
	}
	else {
      activePoint[index].x = xform->pa*__px+xform->pb*__py+xform->pc;
      activePoint[index].y = xform->pd*__px+xform->pe*__py+xform->pf;
      activePoint[index].z = __pz;
    }
    //    activePoint[index].z=z;  // 3d hack does not transform them here

    if (d_g_Flame.symmetryKind != 0.0f && xformIndex < d_g_Flame.numTrans) // does not apply to final xform
    {
        if (d_g_Flame.symmetryKind > 0.0f)
        {
            float rn;
            rn = randFloat(randStates);
            float sina, cosa;
            sincosf(2.0f*PI*floorf(rn*d_g_Flame.symmetryKind)/d_g_Flame.symmetryKind, &sina, &cosa);

            __x = cosa*activePoint[index].x-sina*activePoint[index].y;
            __y = sina*activePoint[index].x+cosa*activePoint[index].y;
            activePoint[index].x = __x;
            activePoint[index].y = __y;
        }
        else
        {
            //pick a random symmetry plane and reflect across it.
            float rn;
            float rn2;
            rn2 = randFloat(randStates);
            rn = randFloat(randStates);
            float sina, cosa;
            sincosf(2.0f*PI*floorf(rn*d_g_Flame.symmetryKind)/d_g_Flame.symmetryKind, &sina, &cosa);

            __x = cosa*activePoint[index].x-sina*activePoint[index].y;
            __y = sina*activePoint[index].x+cosa*activePoint[index].y;
            if (rn2>0.5f)
                __x = -__x;
            activePoint[index].x = __x;
            activePoint[index].y = __y;
        }
    }
    activePoint[index].pal =  pal0 + xform->var_color * (__pal - pal0);
#ifdef JWF_EXTENSIONS	
    activePoint[index].doHide = __doHide;
	activePoint[index].useRgb = __useRgb;
    if(__useRgb) {
	  activePoint[index].colorR = __colorR; 
	  activePoint[index].colorG = __colorG; 
	  activePoint[index].colorB = __colorB; 
	  activePoint[index].colorA = __colorA; 
	}
#endif
}

#ifndef FOR_2D
__device__ void applyRotation(struct point* point, float rotatedViewOffsetx, float rotatedViewOffsety)
{
    point->x += rotatedViewOffsetx;
    point->y += rotatedViewOffsety;
}

__device__ void applyOnlyCamera(struct point* point, float srcX, float srcY, float srcZ, float zdist, float zr)
{
    point->x = srcX / zr;
    point->y = srcY / zr;
	point->z = srcZ;	
}

__device__ void applyDOFAndCamera(struct point* point, float srcX, float srcY, float srcZ, float zdist, float zr, int dofType, float dofScale, float dofFade, float camDOF_10, float rnd1, float rnd2)
{    
    float fade;    
	if (dofFade <= 1.e-6f) {
      fade = 1.0f;
    }
    else if (dofFade >= 1.0f - 1.e-6f) {
      fade = rnd1;
    }
    else {
      fade = rnd2 <= dofFade ? rnd1 : 1.0f;
    }

	float dr = fade * camDOF_10 * zdist * dofScale;

    switch(dofType) {
	  case 0: // BUBBLE
	  default:
	    {
			float a = 2.0f * PI * rnd2;
			float dsina, dcosa;
			sincosf(a, &dsina, &dcosa);
			point->x = (srcX + dr * dcosa) / zr;
			point->y = (srcY + dr * dsina) / zr;
			point->z = srcZ;
			break;
		}
	   case 1: // SINEBLUR
         {
		   float power = 4.2f;
		   float a = 2.0f * PI * rnd2;
		   float dsina, dcosa;
		   sincosf(a, &dsina, &dcosa);

           dr *= (acosf(expf(logf(rnd1) * power) * 2.0f - 1.0f) / PI);

   		   point->x = (srcX + dr * dcosa) / zr;
		   point->y = (srcY + dr * dsina) / zr;
		   point->z = srcZ;
		   break;
         }		 
	}

}

__device__ float clamp(float val, float min, float  max) {
    return fmaxf(min, fminf(max, val));
}

__device__ float smootherstep(float edge0, float edge1, float x) {
    x = clamp((x - edge0) / (edge1 - edge0), 0.0f, 1.0f);
    return x * x * x * (x * (x * 6.f - 15.f) + 10.f);
  }

__device__ void projectJWF(struct point *p, struct CameraViewProperties *properties,
float rnd1, float rnd2)
{
#ifndef JWF_EXTENSIONS	
    float px, py, pz, pw;
    px = properties->matrix[0]*p->x + properties->matrix[4]*p->y + properties->matrix[8]*p->z+ properties->matrix[12];
    py = properties->matrix[1]*p->x + properties->matrix[5]*p->y + properties->matrix[9]*p->z+ properties->matrix[13];
    pz = properties->matrix[2]*p->x + properties->matrix[6]*p->y + properties->matrix[10]*p->z+ properties->matrix[14];
    pw = properties->matrix[3]*p->x + properties->matrix[7]*p->y + properties->matrix[11]*p->z+ properties->matrix[15];

    // handle Apophysis perspective perspective == 0.f ==> Ortho, perspective == 1.f ==> Normal Perspective
    pw  = 1.f + (pw - 1.f) * properties->perspective;

    if (properties->dof > 1.e-6f) {
        float zdist = properties->zpos - pz;
        float t     = rnd1 * 2.f * M_PI_F;
        float dr    = rnd2 * 0.1f * properties->dof * zdist;
        float sina, cosa;
        sincosf(t, &sina, &cosa);

        if (zdist > 0.f) {
            p->x = (px + dr*cosa)/pw;
            p->y = (py + dr*sina)/pw;
            p->z = pz/pw;
        }
        else {
            p->x = px/pw;
            p->y = py/pw;
            p->z = pz/pw;
        }
    }
    else {
        p->x = px/pw;
        p->y = py/pw;
        p->z = pz/pw;
    }
#else 
    float camPointX = properties->matrix[0]*p->x + properties->matrix[4]*p->y + properties->matrix[8]*p->z+ properties->matrix[12];
    float camPointY = properties->matrix[1]*p->x + properties->matrix[5]*p->y + properties->matrix[9]*p->z+ properties->matrix[13];
    float camPointZ = properties->matrix[2]*p->x + properties->matrix[6]*p->y + properties->matrix[10]*p->z+ properties->matrix[14];
	  float camDOF_10 = 0.1 * properties->camDOF; 	  
	  float area = properties->camDOFArea;
      float fade = properties->camDOFArea / 2.25f;
      float areaMinusFade = area - fade;
  	  
      camPointX += properties->camPosX;
      camPointY += properties->camPosY;
      camPointZ += properties->camPosZ;

      float zr = 1.0f - properties->perspective * camPointZ + properties->camPosZ;
      if (zr < 1.e-6f) {
        zr = 1.e-6f;
      }
	  
      if (properties->camDOF > 1.e-6f) {
        if (properties->legacyDOF) {
          float zdist = properties->camDist - camPointZ;
          if (zdist > 0.0f) {
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, rnd1, rnd2);
          }
          else {
            applyOnlyCamera(p, camPointX, camPointY, camPointZ, zdist, zr);
          }
        }
        else {
          float xdist = (camPointX - properties->focusX);
          float ydist = (camPointY - properties->focusY);
          float zdist = (camPointZ - properties->focusZ);

          float dist = powf(xdist * xdist + ydist * ydist + zdist * zdist, 1.0f / properties->camDOFExponent );
          if (dist > area) {
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, rnd1, rnd2);
          }
          else if (dist > areaMinusFade) {
            double scl = smootherstep(0.0f, 1.0f, (dist - areaMinusFade) / fade);
            double sclDist = scl * dist;
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, rnd1, rnd2);
          }
          else {
            applyOnlyCamera(p, camPointX, camPointY, camPointZ, zdist, zr);
          }
        }
      }
      else {
        p->x = camPointX / zr;
        p->y = camPointY / zr;
        p->y = camPointY / zr;
      }
#endif	
}

#else
__device__ 
    void applyRotation(struct point* Point, float cosRotation, float sinRotation)
{
    float x,y;
    x = Point->x-d_g_Flame.center[0];
    y = Point->y-d_g_Flame.center[1];

    Point->x = x*cosRotation - y*sinRotation + d_g_Flame.center[0];
    Point->y = x*sinRotation + y*cosRotation + d_g_Flame.center[1];
}

#endif

__device__
float4 read_image(float4 * image, int length, float index)
{
    float clampedIndex = index - floor(index);
    float scaledIndex = clampedIndex*(float)(length - 1);
    int iLow = floor(scaledIndex);
    int iHigh = ceil(scaledIndex);
    float iFract = scaledIndex - floor(scaledIndex);
    float4 c0 = image[iLow];
    float4 c1 = image[iHigh];
    return make_float4(iFract*c1.x+(1.0f-iFract)*c0.x, 
                        iFract*c1.y+(1.0f-iFract)*c0.y, 
                        iFract*c1.z+(1.0f-iFract)*c0.z, 
                        iFract*c1.w+(1.0f-iFract)*c0.w);
}

extern "C" __global__
void reductionKernal(unsigned* buffer,
                     unsigned length,
                     unsigned * result)
{
    volatile __shared__ unsigned scratch[BLOCKDIM];

    unsigned global_index = blockIdx.x*blockDim.x+threadIdx.x;

    // Perform parallel reduction
    int local_index      = threadIdx.x;
    scratch[local_index] = global_index < length ? buffer[global_index] : 0;
	__syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (local_index < offset) {
            scratch[local_index] += scratch[local_index + offset];
        }
    	__syncthreads();
    }
    if (local_index == 0) {
        result[blockIdx.x] = scratch[0];
    }
}

extern "C" __global__ void iteratePointsKernal(struct VariationListNode *d_g_varUsages,
                                uint *d_g_varUsageIndexes,
                                float *varpars,
                                float *d_g_switchMatrix,
#ifndef FOR_2D
                                struct CameraViewProperties *d_g_Camera,
#endif
                                float4* renderTarget,
                                struct point* points,
                                uint* pointIterations,
                                uint* perThreadRandSeeds,
                                float4* palette,
                                uint numColors,
                                int  paletteStepMode,
                                float epsilon,
                                uint fuseIterations,
                                int xDim,
                                int yDim,
                                uint *startingXform,
                                uint *markedCounts,
                                uint *pixelCounts,
                                uint xformPointPoolSize,
                                uint *permutations,
                                float4 *gradients,
                                uint *shuffle,
                                uint *iterationCount)
{
    __shared__ struct point activePoint[BLOCKDIM];
    __shared__ uint         randStates[WARP_SIZE];
#ifdef PARALLEL_SELECT
    __shared__ uint         rw[2];
#endif
    uint maxR            = d_g_Flame.numTrans - 1;
    uint index           = threadIdx.x; // blockDim.x should be 2 * WARP_SIZE
    uint blockIndex      = blockIdx.y * gridDim.x + blockIdx.x;
    const int ix         = (blockDim.x * blockIndex) + index;
    const uint warpIndex = ix/WARP_SIZE;
    //    const uint whichWarp = warpIndex % WARPS_PER_BLOCK;
    const uint whichWarp = warpIndex & 1;
#ifdef FOR_2D
    float sinRotation, cosRotation;
    sincosf(d_g_Flame.rotation, &sinRotation, &cosRotation);
#endif

    // want to measure the actual number of batches, suspect driver is not executing all batches
    if (blockIndex == 0 && index == 0)
        iterationCount[0] += 1;

    // Iterate some points!
    randStates[index&(WARP_SIZE-1)] = perThreadRandSeeds[ix];

    // want randStates buffers to be populated for entire block before continuing
	__syncthreads();

    uint fromXform = startingXform[warpIndex];
    uint toXform   = 0;
    for (int j = 0; j < NUM_ITERATIONS; j++)
    {
        //Pick xform for this iteration
#ifdef PARALLEL_SELECT
        float w       = randFloatWarp(randStates, whichWarp);
        rw[whichWarp] = 0;
        __syncthreads();

        uint offset = 0;
        uint windex = index & (WARP_SIZE-1);
        while (windex + offset <= maxR) {
            uint lixw = fromXform * d_g_Flame.numTrans + windex + offset;
            if (windex + offset > 0 && w >= d_g_switchMatrix[lixw - 1] && w < d_g_switchMatrix[lixw]) {
                rw[whichWarp] = windex + offset;
            }
            offset += WARP_SIZE;
        }
        __syncthreads();
        uint r = rw[whichWarp];
        toXform = r;
#else
        float w;
        w=randFloatWarp(randStates, whichWarp);
        uint r    = 0;
        while ((r < maxR) && (w > d_g_switchMatrix[fromXform * d_g_Flame.numTrans + r]))
        {
            r++;
        }
        toXform = r;
#endif
        //Now each thread chooses a point at random from the point pool.  This is done to allow each point to have a seperate xform path while retaining SIMD
        uint p               = shuffle[index + NUM_POINTS * j];
        uint fromPointIndex  = fromXform * xformPointPoolSize + NUM_POINTS*blockIndex + p;
        uint toPointIndex    = toXform   * xformPointPoolSize + NUM_POINTS*blockIndex + p;
        uint iterations      = pointIterations[fromPointIndex];
        uint varUsagesIndex  = d_g_varUsageIndexes[r];

        struct point fromPoint = points[fromPointIndex];

        //Iterate the chosen point and store it back to the pool
        iteratePoint(&d_g_varUsages[varUsagesIndex],
                     varpars,
                     &d_g_Xforms[r],
                     r,
                     epsilon,
                     &fromPoint,
                     activePoint,
                     randStates,
                     permutations,
                     gradients);

#ifndef FOR_2D
        if (! isfinite(activePoint[index].x + activePoint[index].y + activePoint[index].z)) {
            // test to add back a random point (ala Flam3) to get Flam3 like images in borderline cases
            activePoint[index].x = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].y = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].z = 0.f; //2.f*randFloat(randStates) - 1.f;
            iterations           = 0;
        }

#else
        if (! isfinite(activePoint[index].x + activePoint[index].y)) {
            // test to add back a random point (ala Flam3) to get Flam3 like images in borderline cases
            activePoint[index].x = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].y = 2.f*randFloat(randStates) - 1.f;
            iterations           = 0;
        }

#endif
        ++iterations;
        struct point toPoint = activePoint[index]; // capture point state before final xform application

        if (iterations >= fuseIterations) { // dont store until fuse for each point is finished
            markedCounts[ix]++;  // keep track of number of iterations that could mark (versus unmarked because of unfused points)

            //Prepare the point for displey.  First the final transformation is applied
            if (d_g_Flame.useFinal) {
                for (uint fIndex = 0; fIndex < d_g_Flame.numFinal; fIndex++) {
                    uint varUsagesIndex  = d_g_varUsageIndexes[d_g_Flame.numTrans + fIndex];
                    struct point tempPoint = activePoint[index];
					float pal_save = activePoint[index].pal;
                    iteratePoint(&d_g_varUsages[varUsagesIndex],
                                 varpars,
                                 &d_g_Xforms[d_g_Flame.numTrans + fIndex],
                                 d_g_Flame.numTrans  + fIndex,
                                 epsilon,
                                 &tempPoint,
                                 activePoint,
                                 randStates,
                                 permutations,
                                 gradients);
								activePoint[index].pal = pal_save; 
                }
            }

#ifdef JWF_EXTENSIONS
         if(!activePoint[index].doHide) {
#endif			

#ifndef FOR_2D
            projectJWF(&activePoint[index], d_g_Camera, RANDFLOAT(), RANDFLOAT());
            applyRotation(&activePoint[index], d_g_Camera->rotatedViewOffsetx, d_g_Camera->rotatedViewOffsety);
#else
            applyRotation( &activePoint[index], cosRotation, sinRotation);
#endif

            //Finally, we randomly jitter the point within a 1/2 pixel radius to obtain antialiasing
            float dr;
            dr = randFloat(randStates);
            dr = expf(d_g_Flame.supersampleWidth*sqrtf(-logf(dr)))-1.0f;
            float rn;
            rn = randFloat(randStates);
            float dtheta = (rn)*2.0f*PI;

            // mark the histogram
            float z = (d_g_Flame.clipToNDC != 0) ? activePoint[index].z : 0.f;
            int x,y;
            float sina, cosa;
            sincosf(dtheta, &sina, &cosa);

            x = floorf((((activePoint[index].x-d_g_Flame.center[0])/d_g_Flame.size[0]+.5f)*(float)xDim)+dr*cosa);
            y = floorf(((-(activePoint[index].y-d_g_Flame.center[1])/d_g_Flame.size[1]+.5f)*(float)yDim)+dr*sina);
            //And render the point to the accumulation buffer
            if ((z >= -1.f) && (z <= 1.f) && (x < xDim)&&(y < yDim)&&(x>=0)&&(y>=0))
            {
                float4 output;
#ifdef JWF_EXTENSIONS
             if(activePoint[index].useRgb) {
               output.x = activePoint[index].colorR;			 
               output.y = activePoint[index].colorG;			 
               output.z = activePoint[index].colorB;			 
               output.w = activePoint[index].colorA;			 
             }
             else {			 
#endif				
                if (paletteStepMode)
                    output = read_imageStepMode(palette, numColors, activePoint[index].pal);
                else
                    output = read_image(palette, numColors, activePoint[index].pal);
                // output = tex1D(texRef,activePoint[threadIdx.x].pal);
				
#ifdef JWF_EXTENSIONS
            } // if(activePoint[index].useRgb) { 
#endif				

#ifdef USE_ATOMICS
                float *ptr = (float *)&(renderTarget[y*xDim+x]);
                atomicAdd(ptr,     output.x*d_g_Xforms[r].opacity);
                atomicAdd(ptr + 1, output.y*d_g_Xforms[r].opacity);
                atomicAdd(ptr + 2, output.z*d_g_Xforms[r].opacity);
                atomicAdd(ptr + 3, output.w*d_g_Xforms[r].opacity);
                atomicAdd(&pixelCounts[y*xDim+x], 1);
#else
                renderTarget[y*xDim+x].x += output.x*d_g_Xforms[r].opacity;
                renderTarget[y*xDim+x].y += output.y*d_g_Xforms[r].opacity;
                renderTarget[y*xDim+x].z += output.z*d_g_Xforms[r].opacity;
                renderTarget[y*xDim+x].w += output.w*d_g_Xforms[r].opacity;
                pixelCounts[y*xDim+x]++;
#endif
            }
#ifdef JWF_EXTENSIONS
         } // if(activePoint[index].doHide==0) {
#endif			
			
        }
        pointIterations[toPointIndex] = iterations;
        points[toPointIndex]          = toPoint;
        fromXform                     = toXform;
        // essential to prevent cross-warp corruption with linear
    	__syncthreads();
        
    }
    startingXform[warpIndex] = toXform;
    perThreadRandSeeds[ix]   = randStates[index&(WARP_SIZE-1)];
}

extern "C" __global__ void postProcessStep1Kernal(
                                float4* renderTarget,
                                float4* accumBuffer,
                                uint xDim,
                                uint yDim,
                                int blocksY,
                                float fuseCompensation)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float k1 = (d_g_Flame.brightness*268.0f)/255.0f;
        float area = fabsf(d_g_Flame.size[0]*d_g_Flame.size[1]);
        float k2 = ((float)(xDim*yDim))/(area*fuseCompensation*((float)(NUM_ITERATIONS))*d_g_Flame.numBatches*32.f*1024.0f*((float)blocksY/32.f));
        float4 rgba = accumBuffer[iy*xDim+ix];
        float a = (k1* logf(1.0f+k2*rgba.w));
        float ls = a/rgba.w;
        rgba.x = ls*rgba.x;
        rgba.y = ls*rgba.y;
        rgba.z = ls*rgba.z;

        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void postProcessStep2Kernal(
                                float4* renderTarget,
                                float4* accumBuffer,
                                uint xDim,
                                uint yDim,
                                int blocksY,
                                float fuseCompensation,
                                float4 adjust)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float k1   = (d_g_Flame.brightness*268.0f)/255.0f;
        float area = fabsf(d_g_Flame.size[0]*d_g_Flame.size[1]);
        float _k2  = ((float)(xDim*yDim))/
                (area*fuseCompensation*((float)(NUM_ITERATIONS))*d_g_Flame.numBatches*32.f*1024.0f*((float)blocksY/32.f));
        float gammaThreshold = d_g_Flame.gammaThreshold;
        float gamma          = d_g_Flame.gamma;
        float alphaGamma     = d_g_Flame.alphaGamma;

        float4 k2   = make_float4(_k2/adjust.x, _k2/adjust.y, _k2/adjust.z, _k2/adjust.w);
        float4 rgba = accumBuffer[iy*xDim+ix];
        
        float4 a = make_float4(k1 * logf(1.0f + k2.x*rgba.w), 
                               k1 * logf(1.0f + k2.y*rgba.w), 
                               k1 * logf(1.0f + k2.z*rgba.w), 
                               k1 * logf(1.0f + k2.w*rgba.w));
                    
                                
        float4 fraction = make_float4(a.x/gammaThreshold, 
                                      a.y/gammaThreshold, 
                                      a.z/gammaThreshold, 
                                      a.w/gammaThreshold);
                                    
        float4 alpha = make_float4(powf(a.x, 1.0f/gamma-1.0f), 
                                    powf(a.y, 1.0f/gamma-1.0f), 
                                    powf(a.z, 1.0f/gamma-1.0f), 
                                    powf(a.w, 1.0f/gamma-1.0f));
                                    
        
        float alphaTx =  (1.f - fraction.x) * a.x * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.x * alpha.x;
        float alphaTy =  (1.f - fraction.y) * a.y * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.y * alpha.y;
        float alphaTz =  (1.f - fraction.z) * a.z * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.z * alpha.z;
        float alphaTw =  (1.f - fraction.w) * a.w * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.w * alpha.w;
                                    
        float4 alphaT   = make_float4(alphaTx, alphaTy, alphaTz, alphaTw);
        
        float4 ls = make_float4(
            d_g_Flame.vibrancy * (a.x < gammaThreshold ? alphaT.x : alpha.x),
            d_g_Flame.vibrancy * (a.y < gammaThreshold ? alphaT.y : alpha.y),
            d_g_Flame.vibrancy * (a.z < gammaThreshold ? alphaT.z : alpha.z),
            d_g_Flame.vibrancy * (a.w < gammaThreshold ? alphaT.w : alpha.w));

        float4 sign = make_float4(
            rgba.x >= 0.f ? 1.f : -1.f,
            rgba.y >= 0.f ? 1.f : -1.f,
            rgba.z >= 0.f ? 1.f : -1.f,
            rgba.w >= 0.f ? 1.f : -1.f);
       
        rgba.x        = ls.x*rgba.x + (1.0f-d_g_Flame.vibrancy)*sign.x*powf(fabsf(rgba.x), 1.0f/gamma);
        rgba.y        = ls.y*rgba.y + (1.0f-d_g_Flame.vibrancy)*sign.y*powf(fabsf(rgba.y), 1.0f/gamma);
        rgba.z        = ls.z*rgba.z + (1.0f-d_g_Flame.vibrancy)*sign.z*powf(fabsf(rgba.z), 1.0f/gamma);
        rgba.w        = ls.w*rgba.w + (1.0f-d_g_Flame.vibrancy)*sign.w*powf(fabsf(rgba.w), 1.0f/gamma);
        
        alpha.x       = powf(a.x, 1.0f/gamma);
        alpha.y       = powf(a.y, 1.0f/gamma);
        alpha.z       = powf(a.z, 1.0f/gamma);
        alpha.w       = powf(a.w, 1.0f/gamma);
        
        alphaT.x      = (1.f - fraction.x) * a.x * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.x * alpha.x;
        alphaT.y      = (1.f - fraction.y) * a.y * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.y * alpha.y;
        alphaT.z      = (1.f - fraction.z) * a.z * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.z * alpha.z;
        alphaT.w      = (1.f - fraction.w) * a.w * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.w * alpha.w;
        
        alpha.x      = fminf((a.x < gammaThreshold ? alphaT.x : alpha.x), 1.f);
        alpha.y      = fminf((a.y < gammaThreshold ? alphaT.y : alpha.y), 1.f);
        alpha.z      = fminf((a.z < gammaThreshold ? alphaT.z : alpha.z), 1.f);
        alpha.w      = fminf((a.w < gammaThreshold ? alphaT.w : alpha.w), 1.f);


        float alphaCw  = powf(a.w, 1.0f/alphaGamma);
        float alphaTCw =  (1.f - fraction.w) * a.w * (powf(gammaThreshold, alphaGamma)/gammaThreshold) + fraction.w * alphaCw;
        alphaCw        = fminf((a.w < gammaThreshold ? alphaTCw : alphaCw), 1.f);


        if (d_g_Flame.highlightPower >= 0.f) {
            rgba = RGBtoHSVHueAdjusted(rgba);
            if (rgba.z > 1.0f)
            {
                //rgba.y /= rgba.z;
                rgba.z = 1.0f;
            }
            rgba = HSVtoRGB(rgba);
        }
        if (isfinite(rgba.x + rgba.y + rgba.z + rgba.w))
        {
            renderTarget[iy*xDim+ix].x=rgba.x+renderTarget[iy*xDim+ix].x*(1.0f-alpha.x);
            renderTarget[iy*xDim+ix].y=rgba.y+renderTarget[iy*xDim+ix].y*(1.0f-alpha.y);
            renderTarget[iy*xDim+ix].z=rgba.z+renderTarget[iy*xDim+ix].z*(1.0f-alpha.z);
            renderTarget[iy*xDim+ix].w=alphaCw;
        }
        else {
            renderTarget[iy*xDim+ix].w=0.f;
        }
    }
}

extern "C" __global__ void colorCurveRGB3ChannelsKernal(float4* accumBuffer,
                                        float* X,
                                        float* A,
                                        float* B,
                                        float* C,
                                        float* D,
                                        uint xDim,
                                        uint yDim,
                                        uint cpCount)

{
    __shared__ float xs[256];
    __shared__ float as[256];
    __shared__ float bs[256];
    __shared__ float cs[256];
    __shared__ float ds[256];

    cpCount    = cpCount > 256 ? 256 : cpCount;
    uint index = threadIdx.x;

    for (uint i = 0; index+i < cpCount; i += blockDim.x) {
        xs[index+i] = X[index+i];
        as[index+i] = A[index+i];
        cs[index+i] = C[index+i];
    }
    for (uint i = 0; index+i < cpCount-1; i += blockDim.x) {
        bs[index+i] = B[index+i];
        ds[index+i] = D[index+i];
    }
	__syncthreads();

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float4 rgba = accumBuffer[iy*xDim+ix];
        // sRGB luma   0.212656 R  0.715158 G  0.072186 B
        // AdobeRGB luma
        float preluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        preluma       /= rgba.w;
        float postluma = curveAdjust(preluma, xs, as, bs, cs, ds, cpCount);

        if (preluma != 0.f) {
            rgba.x = postluma/preluma * rgba.x;
            rgba.y = postluma/preluma * rgba.y;
            rgba.z = postluma/preluma * rgba.z;
            rgba.w = postluma/preluma * rgba.w;
        }
        else {
            rgba.x = postluma;
            rgba.y = postluma;
            rgba.z = postluma;
            rgba.w = postluma;
        }
        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void colorCurveRGBChannelKernal(float4* accumBuffer,
                                        float* X,
                                        float* A,
                                        float* B,
                                        float* C,
                                        float* D,
                                        uint xDim,
                                        uint yDim,
                                        uint cpCount,
                                        uint channel)
{
    __shared__ float xs[256];
    __shared__ float as[256];
    __shared__ float bs[256];
    __shared__ float cs[256];
    __shared__ float ds[256];

    cpCount    = cpCount > 256 ? 256 : cpCount;
    uint index = threadIdx.x;

    for (uint i = 0; index+i < cpCount; i += blockDim.x) {
        xs[index+i] = X[index+i];
        as[index+i] = A[index+i];
        cs[index+i] = C[index+i];
    }
    for (uint i = 0; index+i < cpCount-1; i += blockDim.x) {
        bs[index+i] = B[index+i];
        ds[index+i] = D[index+i];
    }
	__syncthreads();

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float4 rgba = accumBuffer[iy*xDim+ix];

        float preluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        switch (channel) {
            default:
            case 0:
            rgba.x = rgba.w * curveAdjust(rgba.x/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 1:
            rgba.y = rgba.w * curveAdjust(rgba.y/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 2:
            rgba.z = rgba.w * curveAdjust(rgba.z/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 3:
            break;
        }
        float postluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        // maintain same luminance afterwards
        if (preluma != 0)
            rgba.w *= postluma/preluma;
        else
            rgba.w  = postluma;

        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void setBufferKernal(float4* renderTarget, float4 value, uint xDim, uint yDim)
{
    //This kernal simply fills the render target with value
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;

    if ((ix < xDim)&&(iy < yDim))
        renderTarget[iy*xDim+ix] = value;
}

extern "C" __global__ void FlexibleDensityEstimationKernal(float* output, float* input, unsigned int xDim, unsigned int yDim,
                                                            float baseThreshold, int radius)
{
    __shared__ float filterLocal[4*(DENSITY_KERNAL_RADIUS*2+16)*(DENSITY_KERNAL_RADIUS*2+16)];
    unsigned int index    = threadIdx.x;
    const unsigned int ix = blockDim.x*blockIdx.x + index;
    const unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;
    const unsigned int lidx = (blockDim.x+radius*2)*threadIdx.y + index;
    //First, we load up the block of pixels we will be working on into shared memory
    for (unsigned int y = 0; y < blockDim.y+radius*2-threadIdx.y; y += blockDim.y)
    {
        for(unsigned int x = 0; x < blockDim.x+radius*2-index; x += blockDim.x)
        {
            unsigned toIndex4   = 4*(lidx+x+y*(blockDim.x+radius*2));
            //clamped addressing
            unsigned fromIndex4 = 4*(max(min(ix+x-radius,xDim-1),0U)+max(min(iy+y-radius,yDim-1),0U)*xDim);
            filterLocal[toIndex4]     = input[fromIndex4];
            filterLocal[toIndex4 + 1] = input[fromIndex4 + 1];
            filterLocal[toIndex4 + 2] = input[fromIndex4 + 2];
            filterLocal[toIndex4 + 3] = input[fromIndex4 + 3];
        }
    }
    __syncthreads();
    //Next, apply the actual filter
    if ((ix < xDim)&& (iy < yDim))
    {
        unsigned idx = 4*(ix+iy*xDim);
        float pntw = input[idx + 3];
        float sumx = 0.f;
        float sumy = 0.f;
        float sumz = 0.f;
        float sumw = 0.f;

        float count = 0.0f;
        for (int y = 0; y < radius*2+1; y++)
        {
            for (int x = 0; x<radius*2+1; x++)
            {
                int cellIdx   = 4*(lidx+x+y*(blockDim.x+2*radius));
                float invDist = 1.0f/(((float)(x-radius)*(float)(x-radius)+(float)(y-radius)*(float)(y-radius))+1.0f);
                float deviation = fabsf(erff((filterLocal[cellIdx + 3]-pntw)/(sqrtf(8.0f*pntw)+5.0f)));
                if (deviation<=powf(baseThreshold*.9f,sqrtf(1.0f/invDist))*powf(pntw+1.0f,-0.25f))
                {
                    sumx += filterLocal[cellIdx]*invDist;
                    sumy += filterLocal[cellIdx + 1]*invDist;
                    sumz += filterLocal[cellIdx + 2]*invDist;
                    sumw += filterLocal[cellIdx + 3]*invDist;
                    count += invDist;
                }
            }
        }
        sumx/=count;
        sumy/=count;
        sumz/=count;
        sumw/=count;
        //And store the result
        output[idx]     = isfinite(sumx)  ? sumx : 0.f;
        output[idx + 1] =  isfinite(sumy) ? sumy : 0.f;
        output[idx + 2] =  isfinite(sumz) ? sumz : 0.f;
        output[idx + 3] =  isfinite(sumw) ? sumw : 0.f;
    }
}

extern "C" __global__ void RGBA128FtoRGBA32UKernal(uchar4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    //This kernal converts a 32bit per channel floating point image to a 8bit per channel integer image
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_uchar4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*255.0f);
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*255.0f;
                output[iy*xDim+ix] = make_uchar4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    alpha);
            }
            else
                output[iy*xDim+ix]=make_uchar4(0,0,0,0);
            }
        else
        {
        if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
            output[iy*xDim+ix] = make_uchar4(
                fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*255.0f,
                fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*255.0f,
                fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*255.0f,
                255);
        else
            output[iy*xDim+ix]=make_uchar4(0,0,0,255);
        }
    }
}

extern "C" __global__ void RGBA128FtoBGRA32UKernal(uchar4* output, float4* input, uint xDim, uint yDim)
{
    //This kernal converts a 32bit per channel floating point image to a 8bit per channel integer image
    // in BGRA format for little endian Intel with premultiplied alpha
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
        output[iy*xDim+ix] = make_uchar4(
            fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*255.0f, // blue
            fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*255.0f, // green
            fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*255.0f, // red
            255);
        else
            output[iy*xDim+ix]=make_uchar4(0,0,0,255);
    }
}

extern "C" __global__ void RGBA128FtoRGBA64UKernal(ushort4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    //This kernal converts a 32bit per channel floating point image to a 16bit per channel integer image
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_ushort4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f);
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f;
                output[iy*xDim+ix] = make_ushort4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    alpha);
            }
            else
                output[iy*xDim+ix]=make_ushort4(0,0,0,0);
        }
        else
        {
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
                output[iy*xDim+ix] = make_ushort4(
                    fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*65535.0f,
                    fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*65535.0f,
                    fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*65535.0f,
                    65535);
            else
                output[iy*xDim+ix]=make_ushort4(0,0,0,65535);
        }
    }
}

extern "C" __global__ void RGBA128FtoRGBA128FKernal(float4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_float4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f));
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f);
                output[iy*xDim+ix] = make_float4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                alpha);
            }
            else
                output[iy*xDim+ix]=make_float4(0.f,0.f,0.f,0.f);
        }
        else
        {
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
                output[iy*xDim+ix] = make_float4(
                    fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f),
                    fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f),
                    fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f),
                    1.f);
            else
                output[iy*xDim+ix]=make_float4(0.f,0.f,0.f,1.f);
        }
    }
}

extern "C" __global__ void MergeKernal(float4* accum, float4* input, uint xDim, uint yDim)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        accum[iy*xDim+ix].x += input[iy*xDim+ix].x;
        accum[iy*xDim+ix].y += input[iy*xDim+ix].y;
        accum[iy*xDim+ix].z += input[iy*xDim+ix].z;
        accum[iy*xDim+ix].w += input[iy*xDim+ix].w;
    }
}

extern "C" __global__ void readChannelKernel(float* output, float4* input, uint xDim, uint yDim, uint channel)
{
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        switch (channel) {
            default:
            case 0:
            output[iy*xDim+ix] = input[iy*xDim+ix].x;
            break;
            case 1:
            output[iy*xDim+ix] = input[iy*xDim+ix].y;
            break;
            case 2:
            output[iy*xDim+ix] = input[iy*xDim+ix].z;
            break;
            case 3:
            output[iy*xDim+ix] = input[iy*xDim+ix].w;
            break;
        }
    }
}

extern "C" __global__ void writeChannelKernel(float4* output, float* input, uint xDim, uint yDim, uint channel)
{
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        switch (channel) {
            default:
            case 0:
            output[iy*xDim+ix].x = input[iy*xDim+ix];
            break;
            case 1:
            output[iy*xDim+ix].y = input[iy*xDim+ix];
            break;
            case 2:
            output[iy*xDim+ix].z = input[iy*xDim+ix];
            break;
            case 3:
            output[iy*xDim+ix].w = input[iy*xDim+ix];
            break;
        }
    }
}

extern "C" __global__ void writeChannelStripedKernel(float4* output,
                                        float* input,
                                        uint xDim,
                                        uint yDim,
                                        uint channel,
                                        uint supersample)
{
    const uint resampledXdim = xDim / supersample;
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim) && (iy < yDim) && (ix % supersample == 0) && (iy % supersample == 0))
    {
        const uint x = ix / supersample;
        const uint y = iy / supersample;
        switch (channel) {
            default:
            case 0:
            output[y*resampledXdim+x].x = input[iy*xDim+ix];
            break;
            case 1:
            output[y*resampledXdim+x].y = input[iy*xDim+ix];
            break;
            case 2:
            output[y*resampledXdim+x].z = input[iy*xDim+ix];
            break;
            case 3:
            output[y*resampledXdim+x].w = input[iy*xDim+ix];
            break;
        }
    }
}


///////////////////////////////////////////////////////////////////////////////
/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
* OpenCL port & resampling kernels Copyright 2014 Centcom Inc. All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.  Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.   This source code is a "commercial item" as
* that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer  software"  and "commercial computer software
* documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*
* Any use of this source code in individual and commercial software must
* include, in the user documentation and internal comments to the code,
* the above Disclaimer and U.S. Government End Users Notice.
*/

////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////

// Assuming ROW_TILE_W, KERNEL_RADIUS_ALIGNED and dataW
// are multiples of coalescing granularity size,
// all global memory operations are coalesced in convolutionRowGPU()
#define            ROW_TILE_W 128
#define KERNEL_RADIUS_ALIGNED 16

// Assuming COLUMN_TILE_W and dataW are multiples
// of coalescing granularity size, all global memory operations
// are coalesced in convolutionColumnGPU()
#define COLUMN_TILE_W 16
#define COLUMN_TILE_H 48

extern "C" __global__ void convolutionRowsKernel(float *d_Kernel,
                                float *d_Result,
                                float *d_Data,
                                int dataW,
                                int dataH,
                                int KERNEL_RADIUS
){
    //Data cache
    __shared__ float data[KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS_ALIGNED];

    //Current tile and apron limits, relative to row start
    const int         tileStart = blockIdx.x * ROW_TILE_W;
    const int           tileEnd = tileStart + ROW_TILE_W - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataW - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataW - 1);

    //Row start index in d_Data[]
    const int          rowStart = blockIdx.y * dataW;

    //Aligned apron start. Assuming dataW and ROW_TILE_W are multiples
    //of half-warp size, rowStart + apronStartAligned is also a
    //multiple of half-warp size, thus having proper alignment
    //for coalesced d_Data[] read.
    const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;

    const int loadPos = apronStartAligned + threadIdx.x;
    //Set the entire data cache contents
    //Load global memory values, if indices are within the image borders,
    //or initialize with zeroes otherwise
    if(loadPos >= apronStart){
        const int smemPos = loadPos - apronStart;

        // out of bounds set to 0
        // data[smemPos] =
        //     ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ? d_Data[rowStart + loadPos] : 0;

        // reflected at boundary
        // data[smemPos] =
        // loadPos < apronStartClamped ? d_Data[rowStart + apronStartClamped + apronStartClamped - loadPos] :
        //  (loadPos > apronEndClamped ? d_Data[rowStart + apronEndClamped   + apronEndClamped   - loadPos] :
        //     d_Data[rowStart + loadPos]);

        // clamp to border
        data[smemPos] =
            loadPos < apronStartClamped ? d_Data[rowStart + apronStartClamped] :
            (loadPos > apronEndClamped ? d_Data[rowStart + apronEndClamped] :
            d_Data[rowStart + loadPos]);
    }


    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
	__syncthreads();

    const int writePos = tileStart + threadIdx.x;

    //Assuming dataW and ROW_TILE_W are multiples of half-warp size,
    //rowStart + tileStart is also a multiple of half-warp size,
    //thus having proper alignment for coalesced d_Result[] write.
    if(writePos <= tileEndClamped){
        const int smemPos = writePos - apronStart;
        float sum = 0;

        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += data[smemPos + k] * d_Kernel[KERNEL_RADIUS - k];

        d_Result[rowStart + writePos] = isfinite(sum) ? sum : 0.f;
    }
}

extern "C" __global__ void convolutionColumnsKernel(float *d_Kernel,
                                        float *d_Result,
                                        float *d_Data,
                                        int dataW,
                                        int dataH,
                                        int smemStride,
                                        int gmemStride,
                                        int KERNEL_RADIUS
){
    //Data cache
    __shared__ float data[COLUMN_TILE_W * (KERNEL_RADIUS_ALIGNED + COLUMN_TILE_H + KERNEL_RADIUS_ALIGNED)];

    //Current tile and apron limits, in rows
    const int         tileStart = blockIdx.y * COLUMN_TILE_H;
    const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataH - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataH - 1);

    //Current column index
    const int       columnStart = blockIdx.x * COLUMN_TILE_W + threadIdx.x;

    //Shared and global memory indices for current column
    int smemPos    = threadIdx.y * COLUMN_TILE_W + threadIdx.x;
    int gmemPos    = (apronStart + threadIdx.y) * dataW + columnStart;
    int gmemPosMin = columnStart;
    int gmemPosMax = columnStart + dataW * (dataH - 1);

    //Cycle through the entire data cache
    //Load global memory values, if indices are within the image borders,
    //or initialize with zero otherwise
    for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
        // out of bounds set to 0
        //data[smemPos] =
        //    ((y >= apronStartClamped) && (y <= apronEndClamped)) ? d_Data[gmemPos] : 0;

        // reflected at boundary
        // data[smemPos] =
        //     y < apronStartClamped ? d_Data[gmemPosMin + (apronStartClamped - y) * dataW] :
        //    (y > apronEndClamped   ? d_Data[gmemPosMax + (apronEndClamped   - y) * dataW] : d_Data[gmemPos]);

        // clamp to border
        data[smemPos] =
            y < apronStartClamped ? d_Data[gmemPosMin] :
            (y > apronEndClamped   ? d_Data[gmemPosMax] : d_Data[gmemPos]);

        smemPos += smemStride;
        gmemPos += gmemStride;
    }

    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
	__syncthreads();

    //Shared and global memory indices for current column
    smemPos = (threadIdx.y + KERNEL_RADIUS) * COLUMN_TILE_W + threadIdx.x;
    gmemPos = (tileStart + threadIdx.y) * dataW + columnStart;

    //Cycle through the tile body, clamped by image borders
    //Calculate and output the results
    for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
        float sum = 0;

        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += data[smemPos + k * COLUMN_TILE_W] * d_Kernel[KERNEL_RADIUS - k];

        d_Result[gmemPos] = isfinite(sum) ? sum : 0.f;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }
}

