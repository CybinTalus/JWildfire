#include "hip/hip_runtime.h"
/*
Copyright 2008 Steven Brodhead, Jr.
Copyright 2011-2016 Steven Brodhead, Sr., Centcom Inc.

// All rights reserved.
 
//     Fractal Architect Render Engine - a GPU accelerated flame fractal renderer written in C++
//
//     This is free software; you can redistribute it and/or modify it under the terms of the GNU Lesser
//     General Public License as published by the Free Software Foundation; either version 2.1 of the
//     License, or (at your option) any later version.
//
//     This software is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without
//     even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
//     Lesser General Public License for more details.
//
//     You should have received a copy of the GNU Lesser General Public License along with this software;
//     if not, write to the Free Software Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA
//     02110-1301 USA, or see the FSF site: http://www.fsf.org.
*/
#define NUM_ITERATIONS 100
// #define DENSITY_KERNAL_RADIUS 7
#define DENSITY_KERNAL_RADIUS_16KB 7
#define DENSITY_KERNAL_RADIUS_32KB 14
#define DENSITY_KERNAL_RADIUS_48KB 19
#define NUM_FRAMES 160
#define FRAME_RATE 30
#define BITRATE 54000000

#ifndef SUPERSAMPLE_WIDTH
#define SUPERSAMPLE_WIDTH 0.25f
#endif

#ifndef FLAMEDATA_H
#define FLAMEDATA_H

#define MAX_XFORMS 58 // We're limited to 64KB constant memory for compute capacity 1.0.
// All xForms must fit in this.


#define NO_RGBA_CONSTRUCTOR

#define uint  unsigned
#define ulong unsigned long

#define M_PI_F    3.141592653589793f
#define M_PI_2_F  1.5707963267949f
#define M_PI_4_F  0.78539816339745f
#define M_1_PI_F  0.31830988618379f
#define M_2_PI_F  0.63661977236758f

#define JWF_EXTENSIONS

#ifndef RGBA_H
#define RGBA_H

struct  __align__(16) rgba
{
    float r;
    float g;
    float b;
    float a;
};

#endif

#ifdef JWF_EXTENSIONS
__device__ float sqrtf_safe(float x) {
  if (x <= 0.0f)
    return 0.0f;
  else	
    return sqrtf(x);
}

__device__ float lerpf(float a, float b, float p) {
    return a + (b - a) * p;
}

__device__ float blerpf(float c00, float c10, float c01, float c11, float tx, float ty) {
    return lerpf(lerpf(c00, c10, tx), lerpf(c01, c11, tx), ty);
}

//--------------------------------- Noise -----------
// cudaNoise
// Library of common 3D noise functions for CUDA kernels
// https://github.com/covexp/cuda-noise

namespace cudaNoise {

	// Basis functions
	typedef enum {
		BASIS_CHECKER,
		BASIS_DISCRETE,
		BASIS_LINEARVALUE,
		BASIS_FADEDVALUE,
		BASIS_CUBICVALUE,
		BASIS_PERLIN,
		BASIS_SIMPLEX,
		BASIS_WORLEY,
		BASIS_SPOTS
	} basisFunction;

	// Shaping functions
	typedef enum {
		SHAPE_STEP,
		SHAPE_LINEAR,
		SHAPE_QUADRATIC
	} profileShape;

	// Function blending operators
	typedef enum {
		OPERATOR_ADD,
		OPERATOR_AVG,
		OPERATOR_MUL,
		OPERATOR_MAX,
		OPERATOR_MIN
	} repeatOperator;

#define EPSILON 0.000000001f

	// Utility functions

	// Hashing function (used for fast on-device pseudorandom numbers for randomness in noise)
	__device__ unsigned int hash(unsigned int seed)
	{
		seed = (seed + 0x7ed55d16) + (seed << 12);
		seed = (seed ^ 0xc761c23c) ^ (seed >> 19);
		seed = (seed + 0x165667b1) + (seed << 5);
		seed = (seed + 0xd3a2646c) ^ (seed << 9);
		seed = (seed + 0xfd7046c5) + (seed << 3);
		seed = (seed ^ 0xb55a4f09) ^ (seed >> 16);

		return seed;
	}

	// Returns a random integer between [min, max]
	__device__ int randomIntRange(int min, int max, int seed)
	{
		int base = hash(seed);
		base = base % (1 + max - min) + min;

		return base;
	}

	// Returns a random float between [0, 1]
	__device__ float randomFloat(unsigned int seed)
	{
		unsigned int noiseVal = hash(seed);

		return ((float)noiseVal / (float)0xffffffff);
	}

	// Clamps val between [min, max]
	__device__ float clamp(float val, float min, float max)
	{
		if (val < 0.0f)
			return 0.0f;
		else if (val > 1.0f)
			return 1.0f;

		return val;
	}

	// Maps from the signed range [0, 1] to unsigned [-1, 1]
	// NOTE: no clamping
	__device__ float mapToSigned(float input)
	{
		return input * 2.0f - 1.0f;
	}

	// Maps from the unsigned range [-1, 1] to signed [0, 1]
	// NOTE: no clamping
	__device__ float mapToUnsigned(float input)
	{
		return input * 0.5f + 0.5f;
	}

	// Maps from the signed range [0, 1] to unsigned [-1, 1] with clamping
	__device__ float clampToSigned(float input)
	{
		return __saturatef(input) * 2.0f - 1.0f;
	}

	// Maps from the unsigned range [-1, 1] to signed [0, 1] with clamping
	__device__ float clampToUnsigned(float input)
	{
		return __saturatef(input * 0.5f + 0.5f);
	}


	// Random float for a grid coordinate [-1, 1]
	__device__ float randomGrid(int x, int y, int z, int seed = 0)
	{
		return mapToSigned(randomFloat((unsigned int)(x * 1723.0f + y * 93241.0f + z * 149812.0f + 3824.0f + seed)));
	}

	// Random unsigned int for a grid coordinate [0, MAXUINT]
	__device__ unsigned int randomIntGrid(float x, float y, float z, float seed = 0.0f)
	{
		return hash((unsigned int)(x * 1723.0f + y * 93241.0f + z * 149812.0f + 3824 + seed));
	}

	// Random 3D vector as float3 from grid position
	__device__ float3 vectorNoise(int x, int y, int z)
	{
		return make_float3(randomFloat(x * 8231.0f + y * 34612.0f + z * 11836.0f + 19283.0f) * 2.0f - 1.0f,
			randomFloat(x * 1171.0f + y * 9234.0f + z * 992903.0f + 1466.0f) * 2.0f - 1.0f,
			0.0f);
	}

	// Scale 3D vector by scalar value
	__device__ float3 scaleVector(float3 v, float factor)
	{
		return make_float3(v.x * factor, v.y * factor, v.z * factor);
	}

	// Scale 3D vector by nonuniform parameters
	__device__ float3 nonuniformScaleVector(float3 v, float xf, float yf, float zf)
	{
		return make_float3(v.x * xf, v.y * yf, v.z * zf);
	}


	// Adds two 3D vectors
	__device__ float3 addVectors(float3 v, float3 w)
	{
		return make_float3(v.x + w.x, v.y + w.y, v.z + w.z);
	}

	// Dot product between two vectors
	__device__ float dotProduct(float3 u, float3 v)
	{
		return (u.x * v.x + u.y * v.y + u.z * v.z);
	}

	// Device constants for noise

	__device__ __constant__ float gradMap[16][3] = { { 1.0f, 1.0f, 0.0f },{ -1.0f, 1.0f, 0.0f },{ 1.0f, -1.0f, 0.0f },{ -1.0f, -1.0f, 0.0f },
	{ 1.0f, 0.0f, 1.0f },{ -1.0f, 0.0f, 1.0f },{ 1.0f, 0.0f, -1.0f },{ -1.0f, 0.0f, -1.0f },
	{ 0.0f, 1.0f, 1.0f },{ 0.0f, -1.0f, 1.0f },{ 0.0f, 1.0f, -1.0f },{ 0.0f, -1.0f, -1.0f }};

	// Helper functions for noise

	// Linearly interpolate between two float values
	__device__  float lerp(float a, float b, float ratio)
	{
		return a * (1.0f - ratio) + b * ratio;
	}

	// 1D cubic interpolation with four points
	__device__ float cubic(float p0, float p1, float p2, float p3, float x)
	{
		return p1 + 0.5f * x * (p2 - p0 + x * (2.0f * p0 - 5.0f * p1 + 4.0f * p2 - p3 + x * (3.0f * (p1 - p2) + p3 - p0)));
	}

	// Fast gradient function for gradient noise
	__device__ float grad(int hash, float x, float y, float z)
	{
		switch (hash & 0xF)
		{
		case 0x0: return x + y;
		case 0x1: return -x + y;
		case 0x2: return x - y;
		case 0x3: return -x - y;
		case 0x4: return x + z;
		case 0x5: return -x + z;
		case 0x6: return x - z;
		case 0x7: return -x - z;
		case 0x8: return y + z;
		case 0x9: return -y + z;
		case 0xA: return y - z;
		case 0xB: return -y - z;
		case 0xC: return y + x;
		case 0xD: return -y + z;
		case 0xE: return y - x;
		case 0xF: return -y - z;
		default: return 0; // never happens
		}
	}

	// Ken Perlin's fade function for Perlin noise
	__device__ float fade(float t)
	{
		return t * t * t * (t * (t * 6.0f - 15.0f) + 10.0f);     // 6t^5 - 15t^4 + 10t^3
	}

	// Dot product using a float[3] and float parameters
	// NOTE: could be cleaned up
	__device__ float dot(float g[3], float x, float y, float z) {
		return g[0] * x + g[1] * y + g[2] * z;
	}

	// Random value for simplex noise [0, 255]
	__device__ unsigned char calcPerm(int p)
	{
		return (unsigned char)(hash(p));
	}

	// Random value for simplex noise [0, 11]
	__device__ unsigned char calcPerm12(int p)
	{
		return (unsigned char)(hash(p) % 12);
	}

	// Noise functions

	// Simplex noise adapted from Java code by Stefan Gustafson and Peter Eastman
	__device__ float simplexNoise(float3 pos, float scale, int seed)
	{
		float xin = pos.x * scale;
		float yin = pos.y * scale;
		float zin = pos.z * scale;

		// Skewing and unskewing factors for 3 dimensions
		float F3 = 1.0f / 3.0f;
		float G3 = 1.0f / 6.0f;

		float n0, n1, n2, n3; // Noise contributions from the four corners

								// Skew the input space to determine which simplex cell we're in
		float s = (xin + yin + zin)*F3; // Very nice and simple skew factor for 3D
		int i = floorf(xin + s);
		int j = floorf(yin + s);
		int k = floorf(zin + s);
		float t = (i + j + k)*G3;
		float X0 = i - t; // Unskew the cell origin back to (x,y,z) space
		float Y0 = j - t;
		float Z0 = k - t;
		float x0 = xin - X0; // The x,y,z distances from the cell origin
		float y0 = yin - Y0;
		float z0 = zin - Z0;

		// For the 3D case, the simplex shape is a slightly irregular tetrahedron.
		// Determine which simplex we are in.
		int i1, j1, k1; // Offsets for second corner of simplex in (i,j,k) coords
		int i2, j2, k2; // Offsets for third corner of simplex in (i,j,k) coords
		if (x0 >= y0) {
			if (y0 >= z0)
			{
				i1 = 1.0f; j1 = 0.0f; k1 = 0.0f; i2 = 1.0f; j2 = 1.0f; k2 = 0.0f;
			} // X Y Z order
			else if (x0 >= z0) { i1 = 1.0f; j1 = 0.0f; k1 = 0.0f; i2 = 1.0f; j2 = 0.0f; k2 = 1.0f; } // X Z Y order
			else { i1 = 0.0f; j1 = 0.0f; k1 = 1.0f; i2 = 1.0f; j2 = 0.0f; k2 = 1.0f; } // Z X Y order
		}
		else { // x0<y0
			if (y0 < z0) { i1 = 0.0f; j1 = 0.0f; k1 = 1.0f; i2 = 0.0f; j2 = 1; k2 = 1.0f; } // Z Y X order
			else if (x0 < z0) { i1 = 0.0f; j1 = 1.0f; k1 = 0.0f; i2 = 0.0f; j2 = 1.0f; k2 = 1.0f; } // Y Z X order
			else { i1 = 0.0f; j1 = 1.0f; k1 = 0.0f; i2 = 1.0f; j2 = 1.0f; k2 = 0.0f; } // Y X Z order
		}

		// A step of (1,0,0) in (i,j,k) means a step of (1-c,-c,-c) in (x,y,z),
		// a step of (0,1,0) in (i,j,k) means a step of (-c,1-c,-c) in (x,y,z), and
		// a step of (0,0,1) in (i,j,k) means a step of (-c,-c,1-c) in (x,y,z), where
		// c = 1/6.
		float x1 = x0 - i1 + G3; // Offsets for second corner in (x,y,z) coords
		float y1 = y0 - j1 + G3;
		float z1 = z0 - k1 + G3;
		float x2 = x0 - i2 + 2.0f*G3; // Offsets for third corner in (x,y,z) coords
		float y2 = y0 - j2 + 2.0f*G3;
		float z2 = z0 - k2 + 2.0f*G3;
		float x3 = x0 - 1.0f + 3.0f*G3; // Offsets for last corner in (x,y,z) coords
		float y3 = y0 - 1.0f + 3.0f*G3;
		float z3 = z0 - 1.0f + 3.0f*G3;

        int gi0 = calcPerm12(seed + (i * 607495) + (j * 359609) + (k * 654846));
        int gi1 = calcPerm12(seed + (i + i1) * 607495 + (j + j1) * 359609 + (k + k1) * 654846);
        int gi2 = calcPerm12(seed + (i + i2) * 607495 + (j + j2) * 359609 + (k + k2) * 654846);
        int gi3 = calcPerm12(seed + (i + 1) * 607495 + (j + 1) * 359609 + (k + 1) * 654846);

		// Calculate the contribution from the four corners
		float t0 = 0.6f - x0 * x0 - y0 * y0 - z0 * z0;
		if (t0 < 0.0f) n0 = 0.0f;
		else {
			t0 *= t0;
			n0 = t0 * t0 * dot(gradMap[gi0], x0, y0, z0);
		}
		float t1 = 0.6f - x1 * x1 - y1 * y1 - z1 * z1;
		if (t1 < 0.0f) n1 = 0.0f;
		else {
			t1 *= t1;
			n1 = t1 * t1 * dot(gradMap[gi1], x1, y1, z1);
		}
		float t2 = 0.6f - x2 * x2 - y2 * y2 - z2 * z2;
		if (t2 < 0.0f) n2 = 0.0f;
		else {
			t2 *= t2;
			n2 = t2 * t2 * dot(gradMap[gi2], x2, y2, z2);
		}
		float t3 = 0.6f - x3 * x3 - y3 * y3 - z3 * z3;
		if (t3 < 0.0f) n3 = 0.0f;
		else {
			t3 *= t3;
			n3 = t3 * t3 * dot(gradMap[gi3], x3, y3, z3);
		}

		// Add contributions from each corner to get the final noise value.
		// The result is scaled to stay just inside [-1,1]
		return 32.0f*(n0 + n1 + n2 + n3);
	}

	// Checker pattern
	__device__ float checker(float3 pos, float scale, int seed)
	{
		int ix = (int)(pos.x * scale);
		int iy = (int)(pos.y * scale);
		int iz = (int)(pos.z * scale);

		if ((ix + iy + iz) % 2 == 0)
			return 1.0f;

		return -1.0f;
	}

	// Random spots
	__device__ float spots(float3 pos, float scale, int seed, float size, int minNum, int maxNum, float jitter, profileShape shape)
	{
		if (size < EPSILON)
			return 0.0f;

		int ix = (int)(pos.x * scale);
		int iy = (int)(pos.y * scale);
		int iz = (int)(pos.z * scale);

		float u = pos.x - (float)ix;
		float v = pos.y - (float)iy;
		float w = pos.z - (float)iz;

		float val = -1.0f;

		// We need to traverse the entire 3x3x3 neighborhood in case there are spots in neighbors near the edges of the cell
		for (int x = -1; x < 2; x++)
		{
			for (int y = -1; y < 2; y++)
			{
				for (int z = -1; z < 2; z++)
				{
					int numSpots = randomIntRange(minNum, maxNum, seed + (ix + x) * 823746.0f + (iy + y) * 12306.0f + (iz + z) * 823452.0f + 3234874.0f);

					for (int i = 0; i < numSpots; i++)
					{
						float distU = u - x - (randomFloat(seed + (ix + x) * 23784.0f + (iy + y) * 9183.0f + (iz + z) * 23874.0f * i + 27432.0f) * jitter - jitter / 2.0f);
						float distV = v - y - (randomFloat(seed + (ix + x) * 12743.0f + (iy + y) * 45191.0f + (iz + z) * 144421.0f * i + 76671.0f) * jitter - jitter / 2.0f);
						float distW = w - z - (randomFloat(seed + (ix + x) * 82734.0f + (iy + y) * 900213.0f + (iz + z) * 443241.0f * i + 199823.0f) * jitter - jitter / 2.0f);

						float distanceSq = distU * distU + distV * distV + distW * distW;
						float distanceAbs = 0.0f;

						switch (shape)
						{
						case(SHAPE_STEP):
							if (distanceSq < size)
								val = fmaxf(val, 1.0f);
							else
								val = fmaxf(val, -1.0f);
							break;
						case(SHAPE_LINEAR):
							distanceAbs = fabsf(distU) + fabsf(distV) + fabsf(distW);
							val = fmaxf(val, 1.0f - clamp(distanceAbs, 0.0f, size) / size);
							break;
						case(SHAPE_QUADRATIC):
							val = fmaxf(val, 1.0f - clamp(distanceSq, 0.0f, size) / size);
							break;
						}
					}
				}
			}
		}

		return val;
	}

	// Worley cellular noise
	__device__ float worleyNoise(float3 pos, float scale, int seed, float size, int minNum, int maxNum, float jitter)
	{
		if (size < EPSILON)
			return 0.0f;

		int ix = (int)(pos.x * scale);
		int iy = (int)(pos.y * scale);
		int iz = (int)(pos.z * scale);

		float u = pos.x - (float)ix;
		float v = pos.y - (float)iy;
		float w = pos.z - (float)iz;

		float minDist = 1000000.0f;

		// Traverse the whole 3x3 neighborhood looking for the closest feature point
		for (int x = -1; x < 2; x++)
		{
			for (int y = -1; y < 2; y++)
			{
				for (int z = -1; z < 2; z++)
				{
					int numPoints = randomIntRange(minNum, maxNum, seed + (ix + x) * 823746.0f + (iy + y) * 12306.0f + (iz + z) * 67262.0f);

					for (int i = 0; i < numPoints; i++)
					{
						float distU = u - x - (randomFloat(seed + (ix + x) * 23784.0f + (iy + y) * 9183.0f + (iz + z) * 23874.0f * i + 27432.0f) * jitter - jitter / 2.0f);
						float distV = v - y - (randomFloat(seed + (ix + x) * 12743.0f + (iy + y) * 45191.0f + (iz + z) * 144421.0f * i + 76671.0f) * jitter - jitter / 2.0f);
						float distW = w - z - (randomFloat(seed + (ix + x) * 82734.0f + (iy + y) * 900213.0f + (iz + z) * 443241.0f * i + 199823.0f) * jitter - jitter / 2.0f);

						float distanceSq = distU * distU + distV * distV + distW * distW;

						if (distanceSq < minDist)
							minDist = distanceSq;
					}
				}
			}
		}

		return __saturatef(minDist) * 2.0f - 1.0f;
	}

	// Tricubic interpolation
	__device__ float tricubic(int x, int y, int z, float u, float v, float w)
	{
		// interpolate along x first
		float x00 = cubic(randomGrid(x - 1, y - 1, z - 1), randomGrid(x, y - 1, z - 1), randomGrid(x + 1, y - 1, z - 1), randomGrid(x + 2, y - 1, z - 1), u);
		float x01 = cubic(randomGrid(x - 1, y - 1, z), randomGrid(x, y - 1, z), randomGrid(x + 1, y - 1, z), randomGrid(x + 2, y - 1, z), u);
		float x02 = cubic(randomGrid(x - 1, y - 1, z + 1), randomGrid(x, y - 1, z + 1), randomGrid(x + 1, y - 1, z + 1), randomGrid(x + 2, y - 1, z + 1), u);
		float x03 = cubic(randomGrid(x - 1, y - 1, z + 2), randomGrid(x, y - 1, z + 2), randomGrid(x + 1, y - 1, z + 2), randomGrid(x + 2, y - 1, z + 2), u);

		float x10 = cubic(randomGrid(x - 1, y, z - 1), randomGrid(x, y, z - 1), randomGrid(x + 1, y, z - 1), randomGrid(x + 2, y, z - 1), u);
		float x11 = cubic(randomGrid(x - 1, y, z), randomGrid(x, y, z), randomGrid(x + 1, y, z), randomGrid(x + 2, y, z), u);
		float x12 = cubic(randomGrid(x - 1, y, z + 1), randomGrid(x, y, z + 1), randomGrid(x + 1, y, z + 1), randomGrid(x + 2, y, z + 1), u);
		float x13 = cubic(randomGrid(x - 1, y, z + 2), randomGrid(x, y, z + 2), randomGrid(x + 1, y, z + 2), randomGrid(x + 2, y, z + 2), u);

		float x20 = cubic(randomGrid(x - 1, y + 1, z - 1), randomGrid(x, y + 1, z - 1), randomGrid(x + 1, y + 1, z - 1), randomGrid(x + 2, y + 1, z - 1), u);
		float x21 = cubic(randomGrid(x - 1, y + 1, z), randomGrid(x, y + 1, z), randomGrid(x + 1, y + 1, z), randomGrid(x + 2, y + 1, z), u);
		float x22 = cubic(randomGrid(x - 1, y + 1, z + 1), randomGrid(x, y + 1, z + 1), randomGrid(x + 1, y + 1, z + 1), randomGrid(x + 2, y + 1, z + 1), u);
		float x23 = cubic(randomGrid(x - 1, y + 1, z + 2), randomGrid(x, y + 1, z + 2), randomGrid(x + 1, y + 1, z + 2), randomGrid(x + 2, y + 1, z + 2), u);

		float x30 = cubic(randomGrid(x - 1, y + 2, z - 1), randomGrid(x, y + 2, z - 1), randomGrid(x + 1, y + 2, z - 1), randomGrid(x + 2, y + 2, z - 1), u);
		float x31 = cubic(randomGrid(x - 1, y + 2, z), randomGrid(x, y + 2, z), randomGrid(x + 1, y + 2, z), randomGrid(x + 2, y + 2, z), u);
		float x32 = cubic(randomGrid(x - 1, y + 2, z + 1), randomGrid(x, y + 2, z + 1), randomGrid(x + 1, y + 2, z + 1), randomGrid(x + 2, y + 2, z + 1), u);
		float x33 = cubic(randomGrid(x - 1, y + 2, z + 2), randomGrid(x, y + 2, z + 2), randomGrid(x + 1, y + 2, z + 2), randomGrid(x + 2, y + 2, z + 2), u);

		// interpolate along y
		float y0 = cubic(x00, x10, x20, x30, v);
		float y1 = cubic(x01, x11, x21, x31, v);
		float y2 = cubic(x02, x12, x22, x32, v);
		float y3 = cubic(x03, x13, x23, x33, v);

		// interpolate along z
		return cubic(y0, y1, y2, y3, w);
	}

	// Discrete noise (nearest neighbor)
	__device__ float discreteNoise(float3 pos, float scale, int seed)
	{
		int ix = (int)(pos.x * scale);
		int iy = (int)(pos.y * scale);
		int iz = (int)(pos.z * scale);

		return randomGrid(ix, iy, iz, seed);
	}

	// Linear value noise
	__device__ float linearValue(float3 pos, float scale, int seed)
	{
		float fseed = (float)seed;

		int ix = (int)pos.x;
		int iy = (int)pos.y;
		int iz = (int)pos.z;

		float u = pos.x - ix;
		float v = pos.y - iy;
		float w = pos.z - iz;

		// Corner values
		float a000 = randomGrid(ix, iy, iz, fseed);
		float a100 = randomGrid(ix + 1, iy, iz, fseed);
		float a010 = randomGrid(ix, iy + 1, iz, fseed);
		float a110 = randomGrid(ix + 1, iy + 1, iz, fseed);
		float a001 = randomGrid(ix, iy, iz + 1, fseed);
		float a101 = randomGrid(ix + 1, iy, iz + 1, fseed);
		float a011 = randomGrid(ix, iy + 1, iz + 1, fseed);
		float a111 = randomGrid(ix + 1, iy + 1, iz + 1, fseed);

		// Linear interpolation
		float x00 = lerp(a000, a100, u);
		float x10 = lerp(a010, a110, u);
		float x01 = lerp(a001, a101, u);
		float x11 = lerp(a011, a111, u);

		float y0 = lerp(x00, x10, v);
		float y1 = lerp(x01, x11, v);

		return lerp(y0, y1, w);
	}

	// Linear value noise smoothed with Perlin's fade function
	__device__ float fadedValue(float3 pos, float scale, int seed)
	{
		float fseed = (float)seed;

		int ix = (int)(pos.x * scale);
		int iy = (int)(pos.y * scale);
		int iz = (int)(pos.z * scale);

		float u = fade(pos.x - ix);
		float v = fade(pos.y - iy);
		float w = fade(pos.z - iz);

		// Corner values
		float a000 = randomGrid(ix, iy, iz, fseed);
		float a100 = randomGrid(ix + 1, iy, iz, fseed);
		float a010 = randomGrid(ix, iy + 1, iz, fseed);
		float a110 = randomGrid(ix + 1, iy + 1, iz, fseed);
		float a001 = randomGrid(ix, iy, iz + 1, fseed);
		float a101 = randomGrid(ix + 1, iy, iz + 1, fseed);
		float a011 = randomGrid(ix, iy + 1, iz + 1, fseed);
		float a111 = randomGrid(ix + 1, iy + 1, iz + 1, fseed);

		// Linear interpolation
		float x00 = lerp(a000, a100, u);
		float x10 = lerp(a010, a110, u);
		float x01 = lerp(a001, a101, u);
		float x11 = lerp(a011, a111, u);

		float y0 = lerp(x00, x10, v);
		float y1 = lerp(x01, x11, v);

		return lerp(y0, y1, w) / 2.0f * 1.0f;
	}

	// Tricubic interpolated value noise
	__device__ float cubicValue(float3 pos, float scale, int seed)
	{
		pos.x = pos.x * scale;
		pos.y = pos.y * scale;
		pos.z = pos.z * scale;

		int ix = (int)pos.x;
		int iy = (int)pos.y;
		int iz = (int)pos.z;

		float u = pos.x - ix;
		float v = pos.y - iy;
		float w = pos.z - iz;

		return tricubic(ix, iy, iz, u, v, w);
	}

	// Perlin gradient noise
	__device__ float perlinNoise(float3 pos, float scale, int seed)
	{
		float fseed = (float)seed;

		pos.x = pos.x * scale;
		pos.y = pos.y * scale;
		pos.z = pos.z * scale;

		// zero corner integer position
		float ix = floorf(pos.x);
		float iy = floorf(pos.y);
		float iz = floorf(pos.z);

		// current position within unit cube
		pos.x -= ix;
		pos.y -= iy;
		pos.z -= iz;

		// adjust for fade
		float u = fade(pos.x);
		float v = fade(pos.y);
		float w = fade(pos.z);

		// influence values
		float i000 = grad(randomIntGrid(ix, iy, iz, fseed), pos.x, pos.y, pos.z);
		float i100 = grad(randomIntGrid(ix + 1.0f, iy, iz, fseed), pos.x - 1.0f, pos.y, pos.z);
		float i010 = grad(randomIntGrid(ix, iy + 1.0f, iz, fseed), pos.x, pos.y - 1.0f, pos.z);
		float i110 = grad(randomIntGrid(ix + 1.0f, iy + 1.0f, iz, fseed), pos.x - 1.0f, pos.y - 1.0f, pos.z);
		float i001 = grad(randomIntGrid(ix, iy, iz + 1.0f, fseed), pos.x, pos.y, pos.z - 1.0f);
		float i101 = grad(randomIntGrid(ix + 1.0f, iy, iz + 1.0f, fseed), pos.x - 1.0f, pos.y, pos.z - 1.0f);
		float i011 = grad(randomIntGrid(ix, iy + 1.0f, iz + 1.0f, fseed), pos.x, pos.y - 1.0f, pos.z - 1.0f);
		float i111 = grad(randomIntGrid(ix + 1.0f, iy + 1.0f, iz + 1.0f, fseed), pos.x - 1.0f, pos.y - 1.0f, pos.z - 1.0f);

		// interpolation
		float x00 = lerp(i000, i100, u);
		float x10 = lerp(i010, i110, u);
		float x01 = lerp(i001, i101, u);
		float x11 = lerp(i011, i111, u);

		float y0 = lerp(x00, x10, v);
		float y1 = lerp(x01, x11, v);

		float avg = lerp(y0, y1, w);

		return avg;
	}

// Derived noise functions

	// Fast function for fBm using perlin noise
	__device__ float repeaterPerlin(float3 pos, float scale, int seed, int n, float lacunarity, float decay)
	{
		float acc = 0.0f;
		float amp = 1.0f;

		for (int i = 0; i < n; i++)
		{
			acc += perlinNoise(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, (i + 38) * 27389482) * amp;
			scale *= lacunarity;
			amp *= decay;
		}

		return acc;
	}

    // Fast function for fBm using perlin noise
    __device__ float repeaterPerlinBounded(float3 pos, float scale, int seed, int n, float lacunarity, float decay, float threshold)
    {
        float acc = 1.0f;
        float amp = 1.0f;

        for (int i = 0; i < n; i++)
        {
            acc *= 1.0f - __saturatef(0.5f + 0.5f * perlinNoise(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed ^ ((i + 38) * 27389482))) * amp;

            if(acc < threshold)
            {
                return 0.0f;
            }

            scale *= lacunarity;
            amp *= decay;
        }

        return acc;
    }

    // Fast function for fBm using perlin absolute noise
	// Originally called "turbulence", this method takes the absolute value of each octave before adding
	__device__ float repeaterPerlinAbs(float3 pos, float scale, int seed, int n, float lacunarity, float decay)
	{
		float acc = 0.0f;
		float amp = 1.0f;

		for (int i = 0; i < n; i++)
		{
                        acc += fabsf(perlinNoise(make_float3(pos.x * scale, pos.y * scale, pos.z * scale), 1.0f, seed)) * amp;
			scale *= lacunarity;
			amp *= decay;
		}

		// Map the noise back to the standard expected range [-1, 1]
		return mapToSigned(acc);
	}

	// Fast function for fBm using simplex noise
	__device__ float repeaterSimplex(float3 pos, float scale, int seed, int n, float lacunarity, float decay)
	{
		float acc = 0.0f;
		float amp = 1.0f;

		for (int i = 0; i < n; i++)
		{
            acc += simplexNoise(make_float3(pos.x, pos.y, pos.z), scale, seed) * amp * 0.35f;
			scale *= lacunarity;
			amp *= decay;
            seed = seed ^ ((i + 672381) * 200394);
		}

		return acc;
	}

	// Fast function for fBm using simplex absolute noise
	__device__ float repeaterSimplexAbs(float3 pos, float scale, int seed, int n, float lacunarity, float decay)
	{
		float acc = 0.0f;
		float amp = 1.0f;

		for (int i = 0; i < n; i++)
		{
			acc += fabsf(simplexNoise(make_float3(pos.x, pos.y, pos.z), scale, seed)) * amp * 0.35f;
			scale *= lacunarity;
			amp *= decay;
            seed = seed ^ ((i + 198273) * 928374);
		}

		return mapToSigned(acc);
	}

	// Bounded simplex repeater
    __device__ float repeaterSimplexBounded(float3 pos, float scale, int seed, int n, float lacunarity, float decay, float threshold)
    {
        float acc = 1.0f;
        float amp = 1.0f;

        for (int i = 0; i < n; i++)
        {
            float val = __saturatef((simplexNoise(make_float3(pos.x * scale + 32240.7922f, pos.y * scale + 835622.882f, pos.z * scale + 824.371968f), 1.0f, seed) * 0.3f + 0.5f)) * amp;
            acc -= val;

            if(acc < threshold)
            {
                return 0.0f;
            }

            scale *= lacunarity;
            amp *= decay;
        }

        return acc;
    }

    // Generic fBm repeater
	// NOTE: about 10% slower than the dedicated repeater functions
	__device__ float repeater(float3 pos, float scale, int seed, int n, float lacunarity, float decay, basisFunction basis)
	{
		float acc = 0.0f;
		float amp = 1.0f;

		for (int i = 0; i < n; i++)
		{
			switch (basis)
			{
			case(BASIS_CHECKER):
				acc += checker(make_float3(pos.x * scale + 53872.1923f, pos.y * scale + 58334.4081f, pos.z * scale + 9358.34667f), 1.0f, seed) * amp;
				break;
			case(BASIS_DISCRETE):
				acc += discreteNoise(make_float3(pos.x * scale + 7852.53114f, pos.y * scale + 319739.059f, pos.z * scale + 451336.504f), 1.0f, seed) * amp;
				break;
			case(BASIS_LINEARVALUE):
				acc += linearValue(make_float3(pos.x * scale + 940.748139f, pos.y * scale + 10196.4500f, pos.z * scale + 25650.9789f), 1.0f, seed) * amp;
				break;
			case(BASIS_FADEDVALUE):
				acc += fadedValue(make_float3(pos.x * scale + 7683.26428f, pos.y * scale + 2417.78195f, pos.z * scale + 93889.4897f), 1.0f, seed) * amp;
				break;
			case(BASIS_CUBICVALUE):
				acc += cubicValue(make_float3(pos.x * scale + 6546.80178f, pos.y * scale + 14459.4682f, pos.z * scale + 11616.5811f), 1.0f, seed) * amp;
				break;
			case(BASIS_PERLIN):
				acc += perlinNoise(make_float3(pos.x * scale + 1764.66931f, pos.y * scale + 2593.55017f, pos.z * scale + 4813.24412f), 1.0f, seed) * amp;
				break;
			case(BASIS_SIMPLEX):
				acc += simplexNoise(make_float3(pos.x * scale + 7442.93020f, pos.y * scale + 8341.06698f, pos.z * scale + 66848.7870f), 1.0f, seed) * amp;
				break;
			case(BASIS_WORLEY):
				acc += worleyNoise(make_float3(pos.x * scale + 7619.01285f, pos.y * scale + 57209.0681f, pos.z * scale + 1167.91397f), 1.0f, seed, 0.1f, 4, 4, 1.0f) * amp;
				break;
			case(BASIS_SPOTS):
				acc += spots(make_float3(pos.x * scale + 33836.4116f, pos.y * scale + 2242.51045f, pos.z * scale + 6720.07486f), 1.0f, seed, 0.1f, 0, 4, 1.0f, SHAPE_LINEAR) * amp;
				break;
			}

			scale *= lacunarity;
			amp *= decay;
		}

		return acc;
	}

	// Fractal Simplex noise
	// Unlike the repeater function, which calculates a fixed number of noise octaves, the fractal function continues until
	// the feature size is smaller than one pixel
	__device__ float fractalSimplex(float3 pos, float scale, int seed, float du, int n, float lacunarity, float decay)
	{
		float acc = 0.0f;
		float amp = 1.0f;

		float rdu = 1.0f / du;

		for (int i = 0; i < n; i++)
		{
			acc += simplexNoise(make_float3(pos.x * scale + 617.437379f, pos.y * scale + 196410.219f, pos.z * scale + 321280.627f), 1.0f, seed * (i + 1)) * amp;
			scale *= lacunarity;
			amp *= decay;

			if (scale > rdu)
				break;
		}

		return acc;
	}

	// Generic turbulence function
	// Uses a first pass of noise to offset the input vectors for the second pass
	__device__ float turbulence(float3 pos, float scaleIn, float scaleOut, int seed, float strength, basisFunction inFunc, basisFunction outFunc)
	{
		switch (inFunc)
		{
		case(BASIS_CHECKER):
			pos.x += checker(pos, scaleIn, seed ^ 0x34ff8885) * strength;
			pos.y += checker(pos, scaleIn, seed ^ 0x2d03cba3) * strength;
			pos.z += checker(pos, scaleIn, seed ^ 0x5a76fb1b) * strength;
			break;
		case(BASIS_LINEARVALUE):
			pos.x += linearValue(pos, scaleIn, seed ^ 0x5527fdb8) * strength;
			pos.y += linearValue(pos, scaleIn, seed ^ 0x42af1a2e) * strength;
			pos.z += linearValue(pos, scaleIn, seed ^ 0x1482ee8c) * strength;
			break;
		case(BASIS_FADEDVALUE):
			pos.x += fadedValue(pos, scaleIn, seed ^ 0x295590fc) * strength;
			pos.y += fadedValue(pos, scaleIn, seed ^ 0x30731854) * strength;
			pos.z += fadedValue(pos, scaleIn, seed ^ 0x73d2ca4c) * strength;
			break;
		case(BASIS_CUBICVALUE):
			pos.x += cubicValue(pos, scaleIn, seed ^ 0x663a1f09) * strength;
			pos.y += cubicValue(pos, scaleIn, seed ^ 0x429bf56b) * strength;
			pos.z += cubicValue(pos, scaleIn, seed ^ 0x37fa6fe9) * strength;
			break;
		case(BASIS_PERLIN):
			pos.x += perlinNoise(pos, scaleIn, seed ^ 0x74827384) * strength;
			pos.y += perlinNoise(pos, scaleIn, seed ^ 0x10938478) * strength;
			pos.z += perlinNoise(pos, scaleIn, seed ^ 0x62723883) * strength;
			break;
		case(BASIS_SIMPLEX):
			pos.x += simplexNoise(pos, scaleIn, seed ^ 0x47829472) * strength;
			pos.y += simplexNoise(pos, scaleIn, seed ^ 0x58273829) * strength;
			pos.z += simplexNoise(pos, scaleIn, seed ^ 0x10294647) * strength;
			break;
		case(BASIS_WORLEY):
			pos.x += worleyNoise(pos, scaleIn, seed ^ 0x1d96f515, 1.0f, 4, 4, 1.0f) * strength;
			pos.y += worleyNoise(pos, scaleIn, seed ^ 0x4df308f0, 1.0f, 4, 4, 1.0f) * strength;
			pos.z += worleyNoise(pos, scaleIn, seed ^ 0x2b79442a, 1.0f, 4, 4, 1.0f) * strength;
			break;
		}

		switch (outFunc)
		{
		case(BASIS_CHECKER):
			return checker(pos, scaleOut, seed);
		case(BASIS_LINEARVALUE):
			return linearValue(pos, scaleOut, seed);
		case(BASIS_FADEDVALUE):
			return fadedValue(pos, scaleOut, seed);
		case(BASIS_CUBICVALUE):
			return cubicValue(pos, scaleOut, seed);
		case(BASIS_PERLIN):
			return perlinNoise(pos, scaleOut, seed);
		case(BASIS_SIMPLEX):
			return simplexNoise(pos, scaleIn, seed);
		case(BASIS_WORLEY):
			return worleyNoise(pos, scaleIn, seed, 1.0f, 4, 4, 1.0f);
		}

		return 0.0f;
	}

	// Turbulence using repeaters for the first and second pass
	__device__ float repeaterTurbulence(float3 pos, float scaleIn, float scaleOut, int seed, float strength, int n, basisFunction basisIn, basisFunction basisOut)
	{
		pos.x += (repeater(make_float3(pos.x, pos.y, pos.z), scaleIn, seed ^ 0x41728394, n, 2.0f, 0.5f, basisIn)) * strength;
		pos.y += (repeater(make_float3(pos.x, pos.y, pos.z), scaleIn, seed ^ 0x72837263, n, 2.0f, 0.5f, basisIn)) * strength;
		pos.z += (repeater(make_float3(pos.x, pos.y, pos.z), scaleIn, seed ^ 0x26837363, n, 2.0f, 0.5f, basisIn)) * strength;

		return repeater(pos, scaleOut, seed ^ 0x3f821dab, n, 2.0f, 0.5f, basisOut);
	}

} // namespace

//------------- START of JS CODE--------------------------
// vector operations 2D,3D, 4D

#define vec2 float2
#define vec3 float3
#define vec4 float4

 __device__ float2 operator+(const float2 &a, float d) {
   return make_float2(a.x+d, a.y+d);
 }

 __device__ float2 operator-(const float2 &a, float d) {
   return make_float2(a.x-d, a.y-d);
 }
  __device__ float2 operator*(const float2 &a, float d) {
   return make_float2(a.x*d, a.y*d);
 }
  __device__ float2 operator/(const float2 &a, float d) {
   return make_float2(a.x/d, a.y/d);
 }
 __device__ float2 operator+(const float2 &a, const float2 &b) {
   return make_float2(a.x+b.x, a.y+b.y);
 }
 __device__ float2 operator-(const float2 &a, const float2 &b) {
   return make_float2(a.x-b.x, a.y-b.y);
 }
 __device__ float2 operator*(const float2 &a, const float2 &b) {
   return make_float2(a.x*b.x, a.y*b.y);
 }
  __device__ float2 operator/(const float2 &a, const float2 &b) {
   return make_float2(a.x/b.x, a.y/b.y);
 }
 
  __device__ float3 operator+(const float3 &a, float d) {
   return make_float3(a.x+d, a.y+d, a.z+d);
 }
  __device__ float3 operator-(const float3 &a, float d) {
   return make_float3(a.x-d, a.y-d, a.z-d);
 }
  __device__ float3 operator*(const float3 &a, float d) {
   return make_float3(a.x*d, a.y*d, a.z*d);
 }
  __device__ float3 operator/(const float3 &a, float d) {
   return make_float3(a.x/d, a.y/d, a.z/d);
 }
  
__device__ float3 operator+(const float3 &a, const float3 &b) {
   return make_float3(a.x+b.x, a.y+b.y, a.z+b.z  );
 } 
 __device__ float3 operator-(const float3 &a, const float3 &b) {
   return make_float3(a.x-b.x, a.y-b.y, a.z-b.z  );
 }
  __device__ float3 operator*(const float3 &a, const float3 &b) {
   return make_float3(a.x*b.x, a.y*b.y, a.z*b.z  );
 }
  __device__ float3 operator/(const float3 &a, const float3 &b) {
   return make_float3(a.x/b.x, a.y/b.y, a.z/b.z  );
 }
 
   __device__ float4 operator+(const float4 &a, float d) {
   return make_float4(a.x+d, a.y+d, a.z+d, a.w+d);
 }
   __device__ float4 operator-(const float4 &a, float d) {
   return make_float4(a.x-d, a.y-d, a.z-d, a.w-d);
 }
   __device__ float4 operator*(const float4 &a, float d) {
   return make_float4(a.x*d, a.y*d, a.z*d, a.w*d);
 }
   __device__ float4 operator/(const float4 &a, float d) {
   return make_float4(a.x/d, a.y/d, a.z/d, a.w/d);
 }
 
 __device__ float4 operator+(const float4 &a, const float4 &b) {
   return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w  );
 } 
 __device__ float4 operator-(const float4 &a, const float4 &b) {
   return make_float4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w  );
 } 
  __device__ float4 operator*(const float4 &a, const float4 &b) {
   return make_float4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w  );
 }
 __device__ float4 operator/(const float4 &a, const float4 &b) {
   return make_float4(a.x/b.x, a.y/b.y, a.z/b.z, a.w/b.w  );
 } 

__device__ float atan(float n, float d)
{
	return atanf(n/d);
}

__device__ float atan2 (float y, float x)
{
	return atan2f(y,x);
}
__device__ float sqrt(float a)
{
	return sqrtf(a);
}

__device__ float2 sqrt(float2 a)
{
	return make_float2(sqrtf(a.x),sqrtf(a.y));
}

__device__ float3 sqrt(float3 a)
{
	return make_float3( sqrtf(a.x),sqrtf(a.y),sqrtf(a.z) );
}
		
__device__ float4 sqrt(float4 a)
{
	return make_float4( sqrtf(a.x),sqrtf(a.y),sqrtf(a.z),sqrtf(a.w) );
}	

__device__ float pow(float x,float y)
{
	return  powf(x,y);
}
		
__device__ float2 pow(float2 x,float2 y)
{
	float xr= powf(x.x,y.x);
	float yr= powf(x.y,y.y);
	return make_float2(xr,yr);
}

__device__ float3 pow(float3 x,float3 y)
{
	float xr= powf(x.x,y.x);
	float yr= powf(x.y,y.y);
	float zr= powf(x.z,y.z);
	return make_float3(xr,yr,zr);
}
		
__device__ float4 pow(float4 x,float4 y)
{
	float xr= powf(x.x,y.x);
	float yr= powf(x.y,y.y);
	float zr= powf(x.z,y.z);
	float wr= powf(x.w,y.w);
	return make_float4(xr,yr,zr,wr);
}

		
__device__ float2 expf(float2 x)
{
	float xr= expf(x.x);
	float yr= expf(x.y);
	return make_float2(xr,yr);
}

__device__ float3 expf(float3 x)
{
	float xr= expf(x.x);
	float yr= expf(x.y);
	float zr= expf(x.z);
	return make_float3(xr,yr,zr);
}
		
__device__ float4 expf(float4 x)
{
	float xr= expf(x.x);
	float yr= expf(x.y);
	float zr= expf(x.z);
	float wr= expf(x.w);
	return make_float4(xr,yr,zr,wr);
}
		

__device__ float2 exp2f(float2 x)
{
 	float xr= powf(2.0,x.x);
 	float yr= powf(2.0,x.y);
 	return make_float2(xr,yr);
}

__device__ float3 exp2f(float3 x)
{
 	float xr= powf(2.0,x.x);
 	float yr= powf(2.0,x.y);
 	float zr= powf(2.0,x.z);
 	return make_float3(xr,yr,zr);
}

__device__ float4 exp2f(float4 x)
{
 	float xr= powf(2.0,x.x);
 	float yr= powf(2.0,x.y);
 	float zr= powf(2.0,x.z);
 	float wr= powf(2.0,x.w);
 	return make_float4(xr,yr,zr,wr);
}

__device__ float log2(float d) {
	return log2f(d);
}

__device__ float sin(float x)
{
    return sinf(x);
}

__device__ float2 sinf(float2 x)
{
	float v1,v2;
	v1=sinf(x.x);
	v2=sinf(x.y);
	return make_float2(v1,v2);
}

__device__ float2 sin(float2 x)
{
	float v1,v2;
	v1=sinf(x.x);
	v2=sinf(x.y);
	return make_float2(v1,v2);
}

__device__ float3 sin(float3 a)
{
	float v1,v2,v3;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	return make_float3(v1,v2,v3);
}

__device__ float4 sin(float4 a)
{
	float v1,v2,v3,v4;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	v4=sinf(a.w);
	return make_float4(v1,v2,v3,v4);
}

__device__ float4 sinf(float4 a)
{
	float v1,v2,v3,v4;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	v4=sinf(a.w);
	return make_float4(v1,v2,v3,v4);
}
		
__device__ float cos(float x)
{
	return cosf(x);
}

__device__ float2 cosf(float2 x)
{
	float v1,v2;
	v1=cosf(x.x);
	v2=cosf(x.y);
	return make_float2(v1,v2);
}

__device__ float2 cos(float2 x)
{
	float v1,v2;
	v1=cosf(x.x);
	v2=cosf(x.y);
	return make_float2(v1,v2);
}

__device__ float3 cos(float3 a)
{
	float v1,v2,v3;
	v1=cosf(a.x);
	v2=cosf(a.y);
	v3=cosf(a.z);
	return make_float3(v1,v2,v3);
}

__device__ float4 cos(float4 a)
{
	float v1,v2,v3,v4;
	v1=cosf(a.x);
	v2=cosf(a.y);
	v3=cosf(a.z);
	v4=cosf(a.w);
	return make_float4(v1,v2,v3,v4);
}

		
__device__ float length(float2 a)
{
	return sqrtf(a.x*a.x+a.y*a.y);
}

__device__ float length(float3 a)
{
	return sqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ float length(float4 a)
{
	return  sqrtf(a.x*a.x+a.y*a.y+a.z*a.z+ a.w*a.w);
}

__device__ float2 abs(float2 v)
{
	return make_float2(fabsf(v.x),fabsf(v.y));
}

__device__ float3 abs(float3 v)
{
	return make_float3(fabsf(v.x),fabsf(v.y),fabsf(v.z));
}

__device__ float4 abs(float4 v)
{
	return make_float4(fabsf(v.x),fabsf(v.y),fabsf(v.z),fabsf(v.w));
}

__device__ float sign(float v)
{
	return ((v>0.0)?1.0:(v<0.0)?-1.0:0.0);
}

__device__ float2 sign(float2 v)
{
	return make_float2(sign(v.x),sign(v.y));
}
	
	__device__ float3 sign(float3 v)
{
	return make_float3(sign(v.x),sign(v.y),sign(v.z));
}

__device__ float dot(float v1,float v2)
{
	return v1*v2;
}

__device__ float dot(float2 v1,float2 v2)
{
	return v1.x*v2.x + v1.y*v2.y;
}

__device__ float dot(float3 v1,float3 v2)
{
	return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__device__ float dot(float4 v1,float4 v2)
{
	return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z + v1.w*v2.w;
}
__device__ float3 cross(float3 x,float3 y)
{
	float x1,y1,z1;
	x1=x.y*y.z -y.y*x.z;
	y1=x.z*y.x-y.z*x.x;
	z1=x.x*y.y-y.x*x.y;
	return make_float3(x1,y1,z1);
}

__device__ float mod(float x,float y)
{
	return x- y*floorf(x/y);
}

__device__ float2 mod(float2 x,float y)
{
	float x1,y1;
	x1=	x.x- y*floorf(x.x/y);
	y1=	x.y- y*floorf(x.y/y);
	return make_float2(x1,y1);
}

__device__ float2 mod(float2 x,float2 y)
{

	float x1,y1;
	x1=	x.x- y.x*floorf(x.x/y.x);
	y1=	x.y- y.y*floorf(x.y/y.y);
	return make_float2(x1,y1);
}

__device__ float3 mod(float3 x,float y)
{
	float x1,y1,z1;
	x1=	x.x- y*floorf(x.x/y);
	y1=	x.y- y*floorf(x.y/y);
	z1=	x.z- y*floorf(x.z/y);
	return make_float3(x1,y1,z1);
}

__device__ float3 mod(float3 x,float3 y)
{
	float x1,y1,z1;
	x1 = 	x.x- y.x*floorf(x.x/y.x);
	y1 = 	x.y- y.y*floorf(x.y/y.y);
	z1 = 	x.z- y.z*floorf(x.z/y.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 mod(float4 x,float y)
{
	float x1,y1,z1,w1;
	x1=	x.x- y*floorf(x.x/y);
	y1=	x.y- y*floorf(x.y/y);
	z1=	x.z- y*floorf(x.z/y);
	w1=	x.w- y*floorf(x.w/y);
	return make_float4(x1,y1,z1,w1);
}
__device__ float step(float lim, float x)
{
    return (x<lim)?0.0f:1.0f;
}

__device__ float2 step(float  lim, float2 x)
{
	float x1,y1;
	x1=(x.x<lim)?0.0f:1.0f;
	y1=(x.y<lim)?0.0f:1.0;
	return make_float2(x1,y1);
}
		
__device__ float2 step(float2 lim, float2 x)
{
	float x1,y1;
	x1=(x.x<lim.x)?0.0f:1.0f;
	y1=(x.y<lim.y)?0.0f:1.0f;
	return make_float2(x1,y1);
}

__device__ float3 step(float lim, float3 x)
{
	float x1,y1,z1;
	x1=(x.x<lim)?0.0f:1.0f;
	y1=(x.y<lim)?0.0f:1.0f;
	z1=(x.z<lim)?0.0f:1.0f;
	return make_float3(x1,y1,z1);
}
		
__device__ float3 step(float3 lim, float3 x)
{
	float x1,y1,z1;
	x1=(x.x<lim.x)?0.0f:1.0f;
	y1=(x.y<lim.y)?0.0f:1.0f;
	z1=(x.z<lim.z)?0.0f:1.0f;
	return make_float3(x1,y1,z1);
}

__device__ float4 step(float4 lim, float4 x)
{
	float x1,y1,z1,w1;
	x1=(x.x<lim.x)?0.0f:1.0f;
	y1=(x.y<lim.y)?0.0f:1.0f;
	z1=(x.z<lim.z)?0.0f:1.0f;
	w1=(x.w<lim.w)?0.0f:1.0f;
	return make_float4(x1,y1,z1,w1);
}

		
__device__ float2 floorf(float2 v)
{
	return make_float2(floorf(v.x),floorf(v.y));
}

__device__ float3 floorf(float3 v)
{
	return make_float3(floorf(v.x),floorf(v.y),floorf(v.z));
}

__device__ float4 floorf(float4 v)
{
	return make_float4(floorf(v.x),floorf(v.y),floorf(v.z),floorf(v.w));
}

	
		
__device__ float2 truncf(float2 v)
{
	float x1,y1;
	x1=truncf(v.x);
	y1=truncf(v.y);
	return make_float2(x1,y1);
}

__device__ float3 truncf(float3 v)
{
	float x1,y1,z1;
	x1=truncf(v.x);
	y1=truncf(v.y);
	z1=truncf(v.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 truncf(float4 v)
{
	float x1,y1,z1,w1;
	x1=truncf(v.x);
	y1=truncf(v.y);
	z1=truncf(v.z);
	w1=truncf(v.w);
	return make_float4(x1,y1,z1,w1);
}

		
__device__ float2 roundf(float2 v)
{
	float x1,y1;
	x1=roundf(v.x);
	y1=roundf(v.y);
	return make_float2(x1,y1);
}

__device__ float3 roundf(float3 v)
{
	float x1,y1,z1;
	x1=roundf(v.x);
	y1=roundf(v.y);
	z1=roundf(v.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 roundf(float4 v)
{
	float x1,y1,z1,w1;
	x1=roundf(v.x);
	y1=roundf(v.y);
	z1=roundf(v.z);
	w1=roundf(v.w);
	return make_float4(x1,y1,z1,w1);
}
	
		
__device__ float2 ceilf(float2 v)
{
	float x1,y1;
	x1=ceilf(v.x);
	y1=ceilf(v.y);
	return make_float2(x1,y1);
}

__device__ float3 ceilf(float3 v)
{
	float x1,y1,z1;
	x1=ceilf(v.x);
	y1=ceilf(v.y);
	z1=ceilf(v.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 ceilf(float4 v)
{
	float x1,y1,z1,w1;
	x1=ceilf(v.x);
	y1=ceilf(v.y);
	z1=ceilf(v.z);
	w1=ceilf(v.w);
	return make_float4(x1,y1,z1,w1);
}	
__device__ float fract(float x)
{
	return x-floorf(x);
}

__device__ float2 fract(float2 x)
{
	return make_float2(x.x-floorf(x.x),x.y-floorf(x.y));
}

__device__ float3 fract(float3 x)
{
	float xr,yr,zr;
	xr=x.x-floorf(x.x);
	yr=x.y-floorf(x.y);
	zr=x.z-floorf(x.z);
	return make_float3(xr,yr,zr);
}

__device__ float4 fract(float4 x)
{
	float xr,yr,zr,wr;
	xr=x.x-floorf(x.x);
	yr=x.y-floorf(x.y);
	zr=x.z-floorf(x.z);
	wr=x.w-floorf(x.w);
	return make_float4(xr,yr,zr,wr);
}

__device__ float mix(float x,float y, float a)
{
	float z;
	z= (x*(1.0f-a) + y*a);
	return z;
}

__device__ float2 mix(float2 x,float2 y, float a)
{
	float x1,y1;
	x1= (x.x*(1.0f-a) + y.x*a);
	y1= (x.y*(1.0f-a) + y.y*a);
	return make_float2(x1,y1);
}  

__device__ float3 mix(float3 x,float3 y, float a)
{
	return make_float3((x.x*(1.0f-a) + y.x*a), (x.y*(1.0f-a) + y.y*a), (x.z*(1.0f-a) + y.z*a));
}  

__device__ float4 mix(float4 x,float4 y, float a)
{
	float vx,vy,vz,vw;
	vx= (x.x*(1.0f-a) + y.x*a);
	vy= (x.y*(1.0f-a) + y.y*a);
	vz= (x.z*(1.0f-a) + y.z*a);
	vw= (x.w*(1.0f-a) + y.w*a);
	return make_float4(vx,vy,vz,vw);
} 

__device__ float clamp(float val, float min, float  max) {
    return fmaxf(min, fminf(max, val));
}

__device__ float2 clamp(float2 x,float minVal, float maxVal)
{
	float vx,vy;
	vx=fminf(fmaxf(x.x, minVal), maxVal);
	vy=fminf(fmaxf(x.y, minVal), maxVal);
	return make_float2(vx,vy);
}

__device__ float3 clamp(float3 x,float minVal, float maxVal)
{
	float vx,vy,vz;
	vx=fminf(fmaxf(x.x, minVal), maxVal);
	vy=fminf(fmaxf(x.y, minVal), maxVal);
	vz=fminf(fmaxf(x.z, minVal), maxVal);
	return make_float3(vx,vy,vz);
}

__device__ float4 clamp(float4 x,float minVal, float maxVal)
{
	float vx,vy,vz,vw;
	vx=fminf(fmaxf(x.x, minVal), maxVal);
	vy=fminf(fmaxf(x.y, minVal), maxVal);
	vz=fminf(fmaxf(x.z, minVal), maxVal);
	vw=fminf(fmaxf(x.w, minVal), maxVal);
	return make_float4(vx,vy,vz,vw);
}

__device__ float smootherstep(float edge0, float edge1, float x) {
    x = clamp((x - edge0) / (edge1 - edge0), 0.0f, 1.0f);
    return x * x * x * (x * (x * 6.f - 15.f) + 10.f);
 }
  

__device__ float smoothstep(float edge0, float edge1, float x)
{
	float t= clamp((x - edge0) / (edge1 - edge0), 0.0f, 1.0f);
	return t * t * (3.0f - 2.0f * t);
}

__device__ float2 smoothstep(float e0,float e1,float2 x)
{
	return make_float2(smoothstep(e0,e1,x.x),smoothstep(e0,e1,x.y));
}
		
		
__device__ float2 smoothstep(float2 e0,float2 e1, float2 x) 
{

	return make_float2(smoothstep(e0.x,e1.x,x.x),smoothstep(e0.y,e1.y,x.y));
}
__device__ float3 smoothstep(float e0,float  e1, float3 x) 
{
	return make_float3(smoothstep(e0,e1,x.x),smoothstep(e0,e1,x.y),smoothstep(e0,e1,x.z));
}
		
__device__ float3 smoothstep(float3 e0,float3 e1, float3 x) 
{
	return make_float3(smoothstep(e0.x,e1.x,x.x),smoothstep(e0.y,e1.y,x.y),smoothstep(e0.z,e1.z,x.z));
}

__device__ float4 smoothstep(float e0,float e1, float4 x) 
{
	return make_float4(smoothstep(e0,e1,x.x),smoothstep(e0,e1,x.y),smoothstep(e0,e1,x.z),smoothstep(e0,e1,x.w));
}
		
__device__ float4 smoothstep(float4 e0,float4 e1, float4 x) 
{
	return make_float4(smoothstep(e0.x,e1.x,x.x),smoothstep(e0.y,e1.y,x.y),smoothstep(e0.z,e1.z,x.z),smoothstep(e0.w,e1.w,x.w));
}

__device__ float distance(float2 v1,float2 v2)
{
		float2 dif=v1-v2;
		return sqrt(dif.x*dif.x+dif.y*dif.y);
}

__device__ float distance(float3 v1,float3 v2)
{
	float3 dif=v1-v2;
	return sqrt(dif.x*dif.x+dif.y*dif.y + dif.z*dif.z);
}

__device__ float distance(float4 v1,float4 v2)
{
	float4 dif=v1-v2;
	return sqrt(dif.x*dif.x+dif.y*dif.y + dif.z*dif.z + dif.w*dif.w);
}

__device__ float2 normalize(float2 v1)
{
	float x= v1.x/length(v1);
	float y= v1.y/length(v1);
	return make_float2(x,y);
}

__device__ float3 normalize(float3 v1)
{
	float x= v1.x/length(v1);
	float y= v1.y/length(v1);
	float z= v1.z/length(v1);
	return make_float3(x,y,z);
}
		
__device__ float4 normalize(float4 v1)
{
	float x= v1.x/length(v1);
	float y= v1.y/length(v1);
	float z= v1.z/length(v1);
	float w= v1.w/length(v1);
	return make_float4(x,y,z,w);
}


__device__ float2 min(float2 x, float2 y)
{
	float x1,y1;
	x1=fminf(x.x,y.x);
	y1=fminf(x.y,y.y);
	return make_float2(x1,y1);
}
		
__device__ float3 min(float3 x, float3 y)
{
	float x1,y1,z1;
	x1=fminf(x.x,y.x);
	y1=fminf(x.y,y.y);
	z1=fminf(x.z,y.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 min(float4 x, float4 y)
{
	float x1,y1,z1,w1;
	x1=fminf(x.x,y.x);
	y1=fminf(x.y,y.y);
	z1=fminf(x.z,y.z);
	w1=fminf(x.w,y.w);
	return make_float4(x1,y1,z1,w1);
}
		
__device__ float2 min(float2 x, float y)
{
	float x1,y1;
	x1=fminf(x.x,y);
	y1=fminf(x.y,y);
	return make_float2(x1,y1);
}
		
__device__ float3 min(float3 x, float y)
{
	float x1,y1,z1;
	x1=fminf(x.x,y);
	y1=fminf(x.y,y);
	z1=fminf(x.z,y);
	return make_float3(x1,y1,z1);
}

__device__ float4 min(float4 x, float y)
{
	float x1,y1,z1,w1;
	x1=fminf(x.x,y);
	y1=fminf(x.y,y);
	z1=fminf(x.z,y);
	w1=fminf(x.w,y);
	return make_float4(x1,y1,z1,w1);
}


__device__ float2 max(float2 x,float2 y)
{
	float v1,v2;
	v1=fmaxf(x.x,y.x);
	v2=fmaxf(x.y,y.y);
	return make_float2(v1,v2);
}

__device__ float3 max(float3 x,float3 y)
{
	float v1,v2,v3;
	v1=fmaxf(x.x,y.x); 
	v2=fmaxf(x.y,y.y);
	v3=fmaxf(x.z,y.z);
	return make_float3(v1,v2,v3);
}

__device__ float4 max(float4 x,float4 y)
{
	float v1,v2,v3,v4;
	v1=fmaxf(x.x,y.x);
	v2=fmaxf(x.y,y.y);
	v3=fmaxf(x.z,y.z);
	v4=fmaxf(x.w,y.w);
	return make_float4(v1,v2,v3,v4);
}

__device__ float2 fmaxf(float2 x,float y)
{
	float v1,v2;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	return make_float2(v1,v2);
}
__device__ float2 max(float2 x,float y)
{
	float v1,v2;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	return make_float2(v1,v2);
}

__device__ float3 max(float3 x,float y)
{
	float v1,v2,v3;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	v3=fmaxf(x.z,y);
	return make_float3(v1,v2,v3);
}
	
__device__ float4 max(float4 x,float y)
{
	float v1,v2,v3,v4;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	v3=fmaxf(x.z,y);
	v4=fmaxf(x.w,y);
	return make_float4(v1,v2,v3,v4);
}


struct __align__(8) Mat2 {
	 float a00;
	 float a01;
	 float a10;
	 float a11;
};

__device__ void Mat2_Init(Mat2 *m, float v00, float v10, float v01, float v11) {
  m->a00 = v00;
  m->a01 = v01;
  m->a10 = v10;
  m->a11 = v11;
}

__device__ void Mat2_Init(Mat2 *m, float4 v) {
  m->a00 = v.x;
  m->a10 = v.y;
  m->a01 = v.z;
  m->a11 = v.w;
}

__device__ void Mat2_Init(Mat2 *m, float2 v1, float2 v2) {
  m->a00 = v1.x;
  m->a10 = v1.y;
  m->a01 = v2.x;
  m->a11 = v2.y;
}

__device__ float2 times(Mat2 *m, float2 v){
   	return  make_float2(m->a00*v.x + m->a01*v.y , m->a10*v.x + m->a11*v.y); 
}

__device__ void add(Mat2 *m, float v) {
  m->a00 += v;
  m->a10 += v;
  m->a01 += v;
  m->a11 += v;
}

__device__ void minus(Mat2 *m, float v) {
  m->a00 -= v;
  m->a10 -= v;
  m->a01 -= v;
  m->a11 -= v;
}

__device__ void times(Mat2 *m, float v) {
  m->a00 *= v;
  m->a10 *= v;
  m->a01 *= v;
  m->a11 *= v;
}

__device__ void division(Mat2 *m, float v) {
  m->a00 /= v;
  m->a10 /= v;
  m->a01 /= v;
  m->a11 /= v;
}

struct __align__(8) Mat3 {
	 float a00;
	 float a10;
	 float a20;
	 float a01;
	 float a11;
	 float a21;
	 float a02;
	 float a12;
	 float a22;
};

__device__ void Mat3_Init(Mat3 *m, float v00, float v10, float v20, float v01, float v11, float v21, float v02, float v12, float v22 ) {
  m->a00 = v00;
  m->a10 = v10;
  m->a20 = v20;
  m->a01 = v01;
  m->a11 = v11;
  m->a21 = v21;
  m->a02 = v02;
  m->a12 = v12;
  m->a22 = v22;
}

__device__ void Mat3_Init(Mat3 *m, float3 v1, float3 v2, float3 v3) {
  m->a00 = v1.x;
  m->a10 = v1.y;
  m->a20 = v1.z;
  m->a01 = v2.x;
  m->a11 = v2.y;
  m->a21 = v2.z;
  m->a02 = v3.x;
  m->a12 = v3.y;
  m->a22 = v3.z;
}

__device__ float3 times(Mat3 *m, float3 v){
     return make_float3(m->a00*v.x + m->a01*v.y + m->a02*v.z , m->a10*v.x + m->a11*v.y + m->a12*v.z , m->a20*v.x + m->a21*v.y + m->a22*v.z);
   	
}

__device__ void add(Mat3 *m, float v) {
  m->a00 += v;
  m->a10 += v;
  m->a20 += v;
  m->a01 += v;
  m->a11 += v;
  m->a21 += v;
  m->a02 += v;
  m->a12 += v;
  m->a22 += v;
}

__device__ void minus(Mat3 *m, float v) {
  m->a00 -= v;
  m->a10 -= v;
  m->a20 -= v;
  m->a01 -= v;
  m->a11 -= v;
  m->a21 -= v;
  m->a02 -= v;
  m->a12 -= v;
  m->a22 -= v;
}

__device__ void times(Mat3 *m, float v) {
  m->a00 *= v;
  m->a10 *= v;
  m->a20 *= v;
  m->a01 *= v;
  m->a11 *= v;
  m->a21 *= v;
  m->a02 *= v;
  m->a12 *= v;
  m->a22 *= v;
}

__device__ void division(Mat3 *m, float v) {
  m->a00 /= v;
  m->a10 /= v;
  m->a20 /= v;
  m->a01 /= v;
  m->a11 /= v;
  m->a21 /= v;
  m->a02 /= v;
  m->a12 /= v;
  m->a22 /= v;
}

struct __align__(8) Mathc {
	 float a;
	 float b;
	 float c;
	 float d;
	 float e;
	 float f;	 
};

__device__ float2  transfhcf (float2 xy,float a,float b,float c,float d,float e,float f)
{
  float xt=a*xy.x+b*xy.y+c;
  float yt=d*xy.x+e*xy.y+f;
  return make_float2(xt,yt);
}

__device__ float greyscale(int r,int  g,int b)
{
  int lum,red,green,blue;
  red = (r * 0.299);         
  green = (g * 0.587);         
  blue = (b * 0.114);    
  lum = red + green + blue;    
  return (float)lum/255.0f;
}

__device__ int3 dbl2int(float3 theColor)
  	{
  		int red   =  max(0, min(255, (int)floorf(theColor.x * 256.0f)));
  		int green =  max(0, min(255, (int)floorf(theColor.y * 256.0f)));
  		int blue  =  max(0, min(255, (int)floorf(theColor.z * 256.0f)));
  		return make_int3(red,green,blue);
  	}
	
	
__device__ float3  hsv2rgb (float3 c) 
	{
	  float4 K = make_float4(1.0f, 2.0f / 3.0f, 1.0f / 3.0f, 3.0f);
	  float3 p = abs(fract(make_float3(c.x,c.x,c.x)+(make_float3(K.x,K.y,K.z)))*(6.0f)-(make_float3(K.w,K.w,K.w)));
	  return mix(make_float3(K.x,K.x,K.x), clamp(p - make_float3(K.x,K.x,K.x), 0.0f, 1.0f), c.y)*c.z;
	}
//------------- END of JS CODE--------------------------


struct __align__(8) Complex
{
  float per_fix;
  float re;
  float im;
  float save_re;
  float save_im;
};

__device__ void Complex_Init(Complex *c, float Rp, float Ip) {
  c->re = Rp;
  c->im = Ip;
  c->save_re = 0.f;
  c->save_im = 0.f;
  c->per_fix = 0.f;  
}


	
__device__ float Complex_Mag2(Complex *c) {
    return c->re * c->re + c->im * c->im;
}
  
__device__ float Complex_MagInv(Complex *c) {
    float M2 = Complex_Mag2(c);
    return (M2 < 1e-10 ? 1.0f : 1.0f / M2);
}
  
__device__ void Complex_Recip(Complex *c) {
    float mi = Complex_MagInv(c);
    c->re = c->re * mi;
    c->im = -c->im * mi;
}

__device__ void Complex_Dec(Complex *c) {
  c->re -= 1.0f;
}

__device__ void Complex_Inc(Complex *c) {
  c->re += 1.0f;
}

__device__ void Complex_Neg(Complex *c) {
  c->re = -c->re;
  c->im = -c->im;
}
  
__device__ void Complex_Div(Complex *c, Complex *zz) {
  float r2 = c->im * zz->im + c->re * zz->re;
  float i2 = c->im * zz->re - c->re * zz->im;
  float M2 = Complex_MagInv(zz);
  c->re = r2 * M2;
  c->im = i2 * M2;
}
  
__device__ void Complex_Copy(Complex *c, Complex *zz) {
  c->re = zz->re;
  c->im = zz->im;
}
  
__device__ float Complex_Mag2eps(Complex *c) {
    return c->re * c->re + c->im * c->im + 1e-10;
}

__device__ float Complex_Arg(Complex *c) {
  return (c->per_fix + atan2f(c->im, c->re));
}

__device__ void Complex_Log(Complex *c) {
  Complex L_eps;
  Complex_Init(&L_eps, 0.5f * logf(Complex_Mag2eps(c)), Complex_Arg(c));
  Complex_Copy(c, &L_eps);
}

__device__ void Complex_Scale(Complex *c, float mul) {
    c->re = c->re * mul;
    c->im = c->im * mul;
}
  
__device__ void Complex_AtanH(Complex *c) {
    Complex D;
	Complex_Init(&D, c->re, c->im);
    Complex_Dec(&D);
    Complex_Neg(&D);
    Complex_Inc(c);
    Complex_Div(c, &D);
    Complex_Log(c);
    Complex_Scale(c, 0.5f);
}

__device__ void Complex_AcotH(Complex *c) {
   Complex_Recip(c);
   Complex_AtanH(c);
}

__device__ void Complex_Flip(Complex *c) {
    float r2 = c->im;
    float i2 = c->re;
    c->re = r2;
    c->im = i2;
  }
  
__device__ void Complex_Sqr(Complex *c) {
  float r2 = c->re * c->re - c->im * c->im;
  float i2 = 2.f * c->re * c->im;
  c->re = r2;
  c->im = i2;
}  

  
__device__ void Complex_Add(Complex *c, Complex *zz) {
  c->re += zz->re;
  c->im += zz->im;
}

__device__ void Complex_Sub(Complex *c, Complex *zz) {
  c->re -= zz->re;
  c->im -= zz->im;
}


__device__ void Complex_Mul(Complex *c, Complex *zz) {
   if (zz->im == 0.0) {
      Complex_Scale(c, zz->re);
      return;
   }
   float  r2 = c->re * zz->re - c->im * zz->im;
   float  i2 = c->re * zz->im + c->im * zz->re;
   c->re = r2;
   c->im = i2;
}
    
  
__device__ void Complex_One(Complex *c) {
  c->re = 1.0f;
  c->im = 0.0f;
}

__device__ void Complex_Conj(Complex *c) {
  c->im = -c->im;
}


__device__ float Complex_Radius(Complex *c) {
    return hypotf(c->re, c->im);
}

__device__ void Complex_Sqrt(Complex *c) {
  float Rad = Complex_Radius(c);
  float sb = (c->im < 0) ? -1.f : 1.f;
  c->im = sb * sqrtf(0.5f * (Rad - c->re));
  c->re = sqrtf(0.5f * (Rad + c->re));
  if (c->per_fix < 0)
    Complex_Neg(c);
}
  
  
__device__ void Complex_ToP(Complex *c, Complex *dst) {
  Complex_Init(dst, Complex_Radius(c), Complex_Arg(c));
}
  
  
__device__ void Complex_UnP(Complex *c, Complex *dst) {
  Complex_Init(dst, c->re * cosf(c->im), c->re * sinf(c->im));
}  
  
__device__ void Complex_Pow(Complex *c, float exp) {
    if (exp == 0.0f) {
      Complex_One(c);
      return;
    }
    float ex = fabsf(exp);
    if (exp < 0) {
      Complex_Recip(c);
    }
    if (ex == 0.5f) {
      Complex_Sqrt(c);
      return;
    }
    if (ex == 1.0f) {
      return;
    }
    if (ex == 2.0f) {
      Complex_Sqr(c);
      return;
    }
    // In general we need sin, cos etc
    Complex PF;
    Complex_ToP(c, &PF);
    PF.re = powf(PF.re, ex);
    PF.im = PF.im * ex;
	
	Complex PFU;	
	Complex_UnP(&PF, &PFU);	
    Complex_Copy(c, &PFU);
  }
  
 
__device__ void Complex_AsinH(Complex *c) {
  Complex D;
  Complex_Init(&D, c->re, c->im);
  Complex_Sqr(&D);
  Complex_Inc(&D);
  Complex_Pow(&D, 0.5f);
  Complex_Add(c, &D);
  Complex_Log(c);
}

__device__ void Complex_AcosH(Complex *c) {
  Complex D;
  Complex_Init(&D, c->re, c->im);
  Complex_Sqr(&D);
  Complex_Dec(&D);
  Complex_Pow(&D, 0.5f);
  Complex_Add(c, &D);
  Complex_Log(c);
}

__device__ void Complex_AcosecH(Complex *c) {
   Complex_Recip(c);
   Complex_AcosH(c);
}

  
#endif

struct __align__(8) xForm
{
    float a;
    float b;
    float c;
    float d;
    float e;
    float f;
    float pa;
    float pb;
    float pc;
    float pd;
    float pe;
    float pf;
    float color;
    float symmetry;
    float weight;
    float opacity;
    float var_color;
    int   rotates;
#ifdef JWF_EXTENSIONS	
    float yzA;
    float yzB;
    float yzC;
    float yzD;
    float yzE;
    float yzF;
    float yzPa;
    float yzPb;
    float yzPc;
    float yzPd;
    float yzPe;
    float yzPf;
    float zxA;
    float zxB;
    float zxC;
    float zxD;
    float zxE;
    float zxF;
    float zxPa;
    float zxPb;
    float zxPc;
    float zxPd;
    float zxPe;
    float zxPf;
	int useXyz;	
#endif	
};

// each xform has a variable length list of active variations and each variation has its own specific variable sized varpar struct
struct __align__(16) VariationListNode
{                        // all of the lists are concatenated into a single buffer - a separate xformUsageIndex has the offset to the xform's first variation in this list
    uint variationID;    // the numeric value identifying the variation from the variation set - NOTE id of zero is used to signify end of list
    uint varparOffset;   // the offset in varpar union list for this variation's specific varpar struct
    uint enterGroup;     // the state transition that handles entering Pre, Normal, and Post variation groups
};

struct __align__(8) unAnimatedxForm
{
    float a;
    float b;
    float d;
    float e;
    int rotates;
};

struct __align__(16) FlameParams
{
    struct rgba background;
    float center[2];                //{x,y}
    float size[2];                    //size/(scale*zoom)
    float scale;
    float zoom;
    float cam_yaw;
    float cam_pitch;
    float cam_perspective;
    int   clipToNDC;
    float cam_dof;
    float cam_zpos;
    float cam_x;
    float cam_y;
    float cam_z;
    float cam_fov;
    float cam_near;
    float cam_orthowide;
    float hue;
    float numBatches;
    float quality;
    float desiredQuality;
    float rotation;
    float symmetryKind;
    float brightness;
    float gamma;
    float gammaThreshold;
    float alphaGamma;
    float vibrancy;
    unsigned int   numTrans;
    unsigned int   numFinal;
    int   useFinal;
    float supersampleWidth;
    int   frame;
    int   useXaos;
    int   oversample;
    float   highlightPower;
    int    estimatorRadius;			// default 7
    float  estimatorCurve;			// default 0.4
#ifdef JWF_EXTENSIONS
	float cam_roll;
	float cam_bank;
	float cam_xfocus;
	float cam_yfocus;
	float cam_zfocus;
	float cam_xpos;
	float cam_ypos;
	float cam_dist;
	float cam_dof_exponent;
	float cam_dof_area;
	bool legacy_dof;
	int dof_type;
	float dof_scale;
	float dof_fade;
#endif
};


struct Flame //  : public std::enable_shared_from_this<Flame>
{
    struct FlameParams params;
    int                numColors;
    struct xForm      *trans;
    struct xForm      *finals;
    struct rgba       *colorIndex;
    float             *colorLocations;
    float             *switchMatrix;
    //  std::vector<SharedVariationChain> xformVarChains;
    //  std::vector<SharedVariationChain> finalVarChains;
//#ifdef __cplusplus
//    Flame();
//    Flame(int numTrans,int paletteSize, int numFinal);
//    Flame(int numTrans,int paletteSize, int numFinal, int alignment);
//    Flame(const Flame &other);
//    Flame(const Flame &other, int alignment);
//    
//    void Clone(Flame** target);
//    void CloneAligned(Flame** target, int alignment);
//    void deleteChildren();
//    void prepareSwitchMatrix (float *brick);
//    ~Flame();
//#endif
} __attribute__ ((aligned (16)));
#endif

__VARPAR_STRUCT_DECLS__


#define PI 3.141592653589793f

#ifndef FLAM4_KERNAL_CUH
#define FLAM4_KERNAL_CUH

struct __align__(16) point
{
    float x;
    float y;
    float z;
    float pal;
#ifdef JWF_EXTENSIONS
	float colorR, colorG, colorB, colorA;
	bool useRgb;
	bool doHide;
#endif
};
#endif

#ifndef FOR_2D
struct  __align__(16) CameraViewProperties
{
    float matrix[16];
    float yaw;
    float pitch;
    float roll;
    float perspective;
    float dof;
    float zpos;
    float cosRoll;
    float sinRoll;
    float camWidth;
    float camHeight;
    float centerX;
    float centerY;
    int   clipToNDC;
    float rotatedViewOffsetx;
    float rotatedViewOffsety;
#ifdef JWF_EXTENSIONS
	float bank;
	float focusX;
	float focusY;
	float focusZ;
	float camPosX;
	float camPosY;
	float camPosZ;
	float camDist;
	float camDOFExponent;
	float camDOFArea;
	float camDOF;
	bool legacyDOF;
	int dofType;
	float dofScale;
	float dofFade;	
#endif	
};
#endif

// so it can be precompiled as part of the build for syntax checking
#ifndef KERNEL_RUNTIME
#define WARP_SIZE 32
#define NUM_POINTS 64
#endif
#define ADD_EPSILON +epsilon
//#define ADD_EPSILON +1.e-7f
//#define ADD_EPSILON +1.e-10f
//#define ADD_EPSILON

#define WARPS_PER_BLOCK 2
#define BLOCKDIM WARPS_PER_BLOCK*WARP_SIZE

#define RANDFLOAT() randFloat(randStates)
#define RANDINT()   randInt(randStates)

__VARIATION_INDEX_DEFINES__

__constant__ FlameParams d_g_Flame;
__constant__ xForm d_g_Xforms[MAX_XFORMS+MAX_XFORMS];  // has both normal & final xforms

__constant__ unsigned int shift1[4] = {6, 2, 13, 3};
__constant__ unsigned int shift2[4] = {13, 27, 21, 12};
__constant__ unsigned int shift3[4] = {18, 2, 7, 13};
__constant__ unsigned int offset[4] = {4294967294, 4294967288, 4294967280, 4294967168};

texture<uchar4, hipTextureType1D, hipReadModeNormalizedFloat> texRef;

__VARIATION_FUNCTION_PROTOTYPES__


__device__ float sqrf(float x) {
  return x*x;
} 

__device__ unsigned int TausStep(unsigned int z, int S1, int S2, int S3, unsigned int M)
{
    unsigned int b = (((z << S1) ^ z) >> S2);
    return (((z &M) << S3) ^ b);
}

__device__ unsigned int randInt(unsigned int *randStates)
{
    unsigned int index = threadIdx.x;
    randStates[index&(WARP_SIZE-1)] = TausStep(randStates[index&(WARP_SIZE-1)], shift1[index&3], shift2[index&3],shift3[index&3],offset[index&3]);
    return (randStates[(index)&(WARP_SIZE-1)]^randStates[(index+1)&(WARP_SIZE-1)]^randStates[(index+2)&(WARP_SIZE-1)]^randStates[(index+3)&(WARP_SIZE-1)]);
}

__device__ float randFloat(unsigned int *randStates)
//This function returns a random float in [0,1] and updates seed
{
    unsigned int y = randInt(randStates);
    return __int_as_float((y&0x007FFFFF)|0x3F800000)-1.0f;
}

__device__ float randFloatWarp(unsigned int *randStates, uint index)
//This function is a workaround for getting a warp wide rand number
{
    randInt(randStates);
    return __int_as_float((randStates[index]&0x007FFFFF)|0x3F800000)-1.0f;
}

__device__ int linear_range_search(float* X, float x, int n)
{
    int index = 256;
    for (int j = n-1; j >= 0; j--) {
        index = (x <= X[j+1]) ? j : index;
    }
    return index;
}

__device__ float curveAdjust(float x,
        float* X,
        float* A,
        float* B,
        float* C,
        float* D,
        uint cpCount)
{
    int index    = linear_range_search(X, x, cpCount - 1);
    float result = (x <= X[0]) ? A[0] : A[cpCount - 1];

    if (index >= 0 && index < (int)cpCount - 1) {
        float t = x - X[index];
        result = A[index] + B[index]*t + C[index]*t*t + D[index]*t*t*t;
    }
    return result;
}


__device__ float4 RGBtoHSV(float4 color)
{
    float r = color.x;
    float g = color.y;
    float b = color.z;
    float mx = fmaxf(fmaxf(r,g),b);
    float mn = fminf(fminf(r,g),b);
    float h,s,v;
    if (mx == mn)
        h = 0.0f;
    else if (mx == r)
        h = .16666666667f*(g-b)/(mx-mn);
    else if (mx == g)
        h = .16666666667f*(b-r)/(mx-mn)+.33333333f;
    else
        h = .16666666667f*(r-g)/(mx-mn)+.66666667f;
    h = h-floorf(h);
    if (mx == 0.0f)
        s = 0.0f;
    else
        s = (mx-mn)/(mx);
    v = mx;
    if (v > 1.0f) // clamp to 1.f if to high value
        v = 1.0f;
    return make_float4(h,s,v,color.w);
}

__device__ float4 RGBtoHSVHueAdjusted(float4 color)
{
    float r = color.x;
    float g = color.y;
    float b = color.z;
    float mx = fmaxf(fmaxf(r,g),b);
    float mn = fminf(fminf(r,g),b);
    float h,s,v;
    if (mx == mn)
        h = 0.0f;
    else if (mx == r)
        h = .16666666667f*(g-b)/(mx-mn);
    else if (mx == g)
        h = .16666666667f*(b-r)/(mx-mn)+.33333333f;
    else
        h = .16666666667f*(r-g)/(mx-mn)+.66666667f;
    h = h-floorf(h);
    if (mx == 0.0f)
        s = 0.0f;
    else
        s = (mx-mn)/(mx);
    v = mx;
    if (v > 1.0f)
    {
        if (h < .33333333f)
        {
            h += (.16666667f-h)*(1.0f-powf(.75f,v-1.0f));
        }
        else if (h < 0.5f)
        {
            h += (h-0.5f)*(1.0f-powf(.75f,v-1.0f));
        }
        else if (h > 0.8333333f)
        {
            h += (h-0.8333333f)*(1.0f-powf(.75f,v-1.0f));
        }
        //float l = .2126f*r+.7152f*g+.0722f*b;
        //float l = (40.0f*r+20.0f*g+b)/61.0f;
        float l = 0.4f+0.4f*cosf(2.0f*PI*(h-0.16666666667f));
        s = fminf(s*powf(1.0f/v,0.6f*(1.0f-l)),s);
    }
    return make_float4(h,s,v,color.w);
}

__device__ float4 HSVtoRGB(float4 color)
{
    float h = color.x;
    float s = color.y;
    float v = color.z;
    float r,g,b;
    int hi = ((int)floorf(h*6.0f))%6;
    float f = h*6.0f-floorf(h*6.0f);
    float p = v*(1.0f-s);
    float q = v*(1.0f-f*s);
    float t = v*(1.0f-(1.0f-f)*s);
    switch (hi)
    {
        case 0:
        {
            r = v;
            g = t;
            b = p;
        }break;
        case 1:
        {
            r = q;
            g = v;
            b = p;
        }break;
        case 2:
        {
            r = p;
            g = v;
            b = t;
        }break;
        case 3:
        {
            r = p;
            g = q;
            b = v;
        }break;
        case 4:
        {
            r = t;
            g = p;
            b = v;
        }break;
        case 5:
        {
            r = v;
            g = p;
            b = q;
        }break;
    }
    return make_float4(r,g,b,color.w);
}









 

__device__ float4 read_imageStepMode(float4 * image, int length, float index)
{
    float clampedIndex = index - floorf(index);
    float scaledIndex = clampedIndex*(float)(length - 1);
    int iLow = floorf(scaledIndex);
    return image[iLow];
}

__device__ float sinhcoshf(float theta, float* ch)
{
    float expt = expf(theta);
    float exptinv = 1.0f / expt;
    *ch =  (expt + exptinv) * 0.5f;
    return (expt - exptinv) * 0.5f;
}

__VARIATION_FUNCTIONS__


__device__ void iteratePoint(struct VariationListNode *varUsageList,
                float *varpars,
                struct xForm* xform,
                uint xformIndex,
                float epsilon,
                struct point *fromPoint,
                struct point *activePoint,
                unsigned int *randStates,
                uint *permutations,
#ifdef JWF_EXTENSIONS
                float4* palette,
                uint numColors,
#endif
                float4 *gradients)
{
    unsigned int index = threadIdx.x;
    activePoint[index] = *fromPoint;

    float s0 = xform->symmetry;
    float s1 = .5f-.5f*s0;
    float __pal = (activePoint[index].pal+xform->color)*s1+activePoint[index].pal*s0;
    float pal0 = __pal;

     float __x, __y, __z;
	 float __colorR=0.f, __colorG=0.f, __colorB=0.f, __colorA=0.f;
	 bool __doHide = false;
	 bool __useRgb = false;
	
	 if(xform->useXyz) {
		 __x = xform->a*activePoint[index].x+xform->b*activePoint[index].y;
		 __y = xform->d*activePoint[index].x+xform->e*activePoint[index].y;
		 __z = activePoint[index].z;

         float nx, ny, nz;  
         ny = xform->yzA * __y + xform->yzB * __z;
         nz = xform->yzD * __y + xform->yzE * __z;
         __y = ny;
         __z = nz;

         nx = xform->zxA * __x + xform->zxB * __z;
         nz = xform->zxD * __x + xform->zxE * __z;
         __x = nx;
         __z = nz;
		 
		 __x += xform->c + xform->zxC;
		 __y += xform->f + xform->yzC;
		 __z += xform->yzF + xform->zxF;
	 }
	 else {
		 __x = xform->a*activePoint[index].x+xform->b*activePoint[index].y+xform->c;
		 __y = xform->d*activePoint[index].x+xform->e*activePoint[index].y+xform->f;
		 __z = activePoint[index].z; // 3d hack does not transform them here
	 }

	float __r2, __r, __rinv, __phi, __theta;
    float __px = __x;  // note that enterGroup action will handle resetting these to zero -- also works correctly for xforms with NO variations set
    float __py = __y;
    float __pz = __z;

	float __x0 = __x;
	float __y0 = __y;
	float __z0 = __z;
	
	bool __was_pre = 0;


			__r2 = __x * __x + __y * __y;
			__r = sqrtf(__r2);
			__rinv = 1.0f / __r;

			__phi = atan2f(__x, __y);
			__theta = .5f*PI - __phi;
			if (__theta > PI)
				__theta -= 2.0f*PI;


     __px = 0.f;
     __py = 0.f;
     __pz = 0.f;


    uint varIndex = 0;
    while ((varIndex = varUsageList->variationID) != 0) {
        float *varparCluster = &varpars[varUsageList->varparOffset];
        switch (varIndex) {
            //Now apply the Variations
            __VARIATION_SWITCH_CASES__
            default:
            break;
        }
        varUsageList++;
    }

    if(xform->useXyz) {
      float px = xform->pa*__px+xform->pb*__py;
      float py = xform->pd*__px+xform->pe*__py;
      float pz = __pz;
	  
	  float nx, ny, nz;
	    
      ny = xform->yzPa * py + xform->yzPb * pz;
      nz = xform->yzPd * py + xform->yzPe * pz;
      py = ny;
      pz = nz;
	 
	  nx = xform->zxPa * px + xform->zxPb * pz;
      nz = xform->zxPd * px + xform->zxPe * pz;
      px = nx;
      pz = nz;
	  
      activePoint[index].x = px + xform->pc + xform->zxPc;
      activePoint[index].y = py + xform->pf + xform->yzPc;
      activePoint[index].z = pz + xform->yzPf + xform->zxPf;
	 
	}
	else {
      activePoint[index].x = xform->pa*__px+xform->pb*__py+xform->pc;
      activePoint[index].y = xform->pd*__px+xform->pe*__py+xform->pf;
      activePoint[index].z = __pz;
    }
    //    activePoint[index].z=z;  // 3d hack does not transform them here

    if (d_g_Flame.symmetryKind != 0.0f && xformIndex < d_g_Flame.numTrans) // does not apply to final xform
    {
        if (d_g_Flame.symmetryKind > 0.0f)
        {
            float rn;
            rn = randFloat(randStates);
            float sina, cosa;
            sincosf(2.0f*PI*floorf(rn*d_g_Flame.symmetryKind)/d_g_Flame.symmetryKind, &sina, &cosa);

            __x = cosa*activePoint[index].x-sina*activePoint[index].y;
            __y = sina*activePoint[index].x+cosa*activePoint[index].y;
            activePoint[index].x = __x;
            activePoint[index].y = __y;
        }
        else
        {
            //pick a random symmetry plane and reflect across it.
            float rn;
            float rn2;
            rn2 = randFloat(randStates);
            rn = randFloat(randStates);
            float sina, cosa;
            sincosf(2.0f*PI*floorf(rn*d_g_Flame.symmetryKind)/d_g_Flame.symmetryKind, &sina, &cosa);

            __x = cosa*activePoint[index].x-sina*activePoint[index].y;
            __y = sina*activePoint[index].x+cosa*activePoint[index].y;
            if (rn2>0.5f)
                __x = -__x;
            activePoint[index].x = __x;
            activePoint[index].y = __y;
        }
    }
    activePoint[index].pal =  pal0 + xform->var_color * (__pal - pal0);
#ifdef JWF_EXTENSIONS	
    activePoint[index].doHide = __doHide;
	activePoint[index].useRgb = __useRgb;
    if(__useRgb) {
	  activePoint[index].colorR = __colorR; 
	  activePoint[index].colorG = __colorG; 
	  activePoint[index].colorB = __colorB; 
	  activePoint[index].colorA = __colorA; 
	}
#endif
}

#ifndef FOR_2D
__device__ void applyRotation(struct point* point, float rotatedViewOffsetx, float rotatedViewOffsety)
{
    point->x += rotatedViewOffsetx;
    point->y += rotatedViewOffsety;
}

__device__ void applyOnlyCamera(struct point* point, float srcX, float srcY, float srcZ, float zdist, float zr)
{
    point->x = srcX / zr;
    point->y = srcY / zr;
	point->z = srcZ;	
}

__device__ void applyDOFAndCamera(struct point* point, float srcX, float srcY, float srcZ, float zdist, float zr, int dofType, float dofScale, float dofFade, float camDOF_10, float rnd1, float rnd2)
{    
    float fade;    
	if (dofFade <= 1.e-6f) {
      fade = 1.0f;
    }
    else if (dofFade >= 1.0f - 1.e-6f) {
      fade = rnd1;
    }
    else {
      fade = rnd2 <= dofFade ? rnd1 : 1.0f;
    }

	float dr = fade * camDOF_10 * zdist * dofScale;

    switch(dofType) {
	  case 0: // BUBBLE
	  default:
	    {
			float a = 2.0f * PI * rnd2;
			float dsina, dcosa;
			sincosf(a, &dsina, &dcosa);
			point->x = (srcX + dr * dcosa) / zr;
			point->y = (srcY + dr * dsina) / zr;
			point->z = srcZ;
			break;
		}
	   case 1: // SINEBLUR
         {
		   float power = 4.2f;
		   float a = 2.0f * PI * rnd2;
		   float dsina, dcosa;
		   sincosf(a, &dsina, &dcosa);

           dr *= (acosf(expf(logf(rnd1) * power) * 2.0f - 1.0f) / PI);

   		   point->x = (srcX + dr * dcosa) / zr;
		   point->y = (srcY + dr * dsina) / zr;
		   point->z = srcZ;
		   break;
         }		 
	}

}



__device__ void projectJWF(struct point *p, struct CameraViewProperties *properties,
float rnd1, float rnd2)
{
#ifndef JWF_EXTENSIONS	
    float px, py, pz, pw;
    px = properties->matrix[0]*p->x + properties->matrix[4]*p->y + properties->matrix[8]*p->z+ properties->matrix[12];
    py = properties->matrix[1]*p->x + properties->matrix[5]*p->y + properties->matrix[9]*p->z+ properties->matrix[13];
    pz = properties->matrix[2]*p->x + properties->matrix[6]*p->y + properties->matrix[10]*p->z+ properties->matrix[14];
    pw = properties->matrix[3]*p->x + properties->matrix[7]*p->y + properties->matrix[11]*p->z+ properties->matrix[15];

    // handle Apophysis perspective perspective == 0.f ==> Ortho, perspective == 1.f ==> Normal Perspective
    pw  = 1.f + (pw - 1.f) * properties->perspective;

    if (properties->dof > 1.e-6f) {
        float zdist = properties->zpos - pz;
        float t     = rnd1 * 2.f * M_PI_F;
        float dr    = rnd2 * 0.1f * properties->dof * zdist;
        float sina, cosa;
        sincosf(t, &sina, &cosa);

        if (zdist > 0.f) {
            p->x = (px + dr*cosa)/pw;
            p->y = (py + dr*sina)/pw;
            p->z = pz/pw;
        }
        else {
            p->x = px/pw;
            p->y = py/pw;
            p->z = pz/pw;
        }
    }
    else {
        p->x = px/pw;
        p->y = py/pw;
        p->z = pz/pw;
    }
#else 
    float camPointX = properties->matrix[0]*p->x + properties->matrix[4]*p->y + properties->matrix[8]*p->z+ properties->matrix[12];
    float camPointY = properties->matrix[1]*p->x + properties->matrix[5]*p->y + properties->matrix[9]*p->z+ properties->matrix[13];
    float camPointZ = properties->matrix[2]*p->x + properties->matrix[6]*p->y + properties->matrix[10]*p->z+ properties->matrix[14];
	  float camDOF_10 = 0.1 * properties->camDOF; 	  
	  float area = properties->camDOFArea;
      float fade = properties->camDOFArea / 2.25f;
      float areaMinusFade = area - fade;
  	  
      camPointX += properties->camPosX;
      camPointY += properties->camPosY;
      camPointZ += properties->camPosZ;

      float zr = 1.0f - properties->perspective * camPointZ + properties->camPosZ;
      if (zr < 1.e-6f) {
        zr = 1.e-6f;
      }
	  
      if (properties->camDOF > 1.e-6f) {
        if (properties->legacyDOF) {
          float zdist = properties->camDist - camPointZ;
          if (zdist > 0.0f) {
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, rnd1, rnd2);
          }
          else {
            applyOnlyCamera(p, camPointX, camPointY, camPointZ, zdist, zr);
          }
        }
        else {
          float xdist = (camPointX - properties->focusX);
          float ydist = (camPointY - properties->focusY);
          float zdist = (camPointZ - properties->focusZ);

          float dist = powf(xdist * xdist + ydist * ydist + zdist * zdist, 1.0f / properties->camDOFExponent );
          if (dist > area) {
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, rnd1, rnd2);
          }
          else if (dist > areaMinusFade) {
            double scl = smootherstep(0.0f, 1.0f, (dist - areaMinusFade) / fade);
            double sclDist = scl * dist;
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, rnd1, rnd2);
          }
          else {
            applyOnlyCamera(p, camPointX, camPointY, camPointZ, zdist, zr);
          }
        }
      }
      else {
        p->x = camPointX / zr;
        p->y = camPointY / zr;
        p->y = camPointY / zr;
      }
#endif	
}

#else
__device__ 
    void applyRotation(struct point* Point, float cosRotation, float sinRotation)
{
    float x,y;
    x = Point->x-d_g_Flame.center[0];
    y = Point->y-d_g_Flame.center[1];

    Point->x = x*cosRotation - y*sinRotation + d_g_Flame.center[0];
    Point->y = x*sinRotation + y*cosRotation + d_g_Flame.center[1];
}

#endif

__device__
float4 read_image(float4 * image, int length, float index)
{
    float clampedIndex = index - floor(index);
    float scaledIndex = clampedIndex*(float)(length - 1);
    int iLow = floor(scaledIndex);
    int iHigh = ceil(scaledIndex);
    float iFract = scaledIndex - floor(scaledIndex);
    float4 c0 = image[iLow];
    float4 c1 = image[iHigh];
    return make_float4(iFract*c1.x+(1.0f-iFract)*c0.x, 
                        iFract*c1.y+(1.0f-iFract)*c0.y, 
                        iFract*c1.z+(1.0f-iFract)*c0.z, 
                        iFract*c1.w+(1.0f-iFract)*c0.w);
}

extern "C" __global__
void reductionKernal(unsigned* buffer,
                     unsigned length,
                     unsigned * result)
{
    volatile __shared__ unsigned scratch[BLOCKDIM];

    unsigned global_index = blockIdx.x*blockDim.x+threadIdx.x;

    // Perform parallel reduction
    int local_index      = threadIdx.x;
    scratch[local_index] = global_index < length ? buffer[global_index] : 0;
	__syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (local_index < offset) {
            scratch[local_index] += scratch[local_index + offset];
        }
    	__syncthreads();
    }
    if (local_index == 0) {
        result[blockIdx.x] = scratch[0];
    }
}

extern "C" __global__ void iteratePointsKernal(struct VariationListNode *d_g_varUsages,
                                uint *d_g_varUsageIndexes,
                                float *varpars,
                                float *d_g_switchMatrix,
#ifndef FOR_2D
                                struct CameraViewProperties *d_g_Camera,
#endif
                                float4* renderTarget,
                                struct point* points,
                                uint* pointIterations,
                                uint* perThreadRandSeeds,
                                float4* palette,
                                uint numColors,
                                int  paletteStepMode,
                                float epsilon,
                                uint fuseIterations,
                                int xDim,
                                int yDim,
                                uint *startingXform,
                                uint *markedCounts,
                                uint *pixelCounts,
                                uint xformPointPoolSize,
                                uint *permutations,
                                float4 *gradients,
                                uint *shuffle,
                                uint *iterationCount)
{
    __shared__ struct point activePoint[BLOCKDIM];
    __shared__ uint         randStates[WARP_SIZE];
#ifdef PARALLEL_SELECT
    __shared__ uint         rw[2];
#endif
    uint maxR            = d_g_Flame.numTrans - 1;
    uint index           = threadIdx.x; // blockDim.x should be 2 * WARP_SIZE
    uint blockIndex      = blockIdx.y * gridDim.x + blockIdx.x;
    const int ix         = (blockDim.x * blockIndex) + index;
    const uint warpIndex = ix/WARP_SIZE;
    //    const uint whichWarp = warpIndex % WARPS_PER_BLOCK;
    const uint whichWarp = warpIndex & 1;
#ifdef FOR_2D
    float sinRotation, cosRotation;
    sincosf(d_g_Flame.rotation, &sinRotation, &cosRotation);
#endif

    // want to measure the actual number of batches, suspect driver is not executing all batches
    if (blockIndex == 0 && index == 0)
        iterationCount[0] += 1;

    // Iterate some points!
    randStates[index&(WARP_SIZE-1)] = perThreadRandSeeds[ix];

    // want randStates buffers to be populated for entire block before continuing
	__syncthreads();

    uint fromXform = startingXform[warpIndex];
    uint toXform   = 0;
	
	
    for (int j = 0; j < NUM_ITERATIONS; j++)
    {
        //Pick xform for this iteration
#ifdef PARALLEL_SELECT
        float w       = randFloatWarp(randStates, whichWarp);
        rw[whichWarp] = 0;
        __syncthreads();

        uint offset = 0;
        uint windex = index & (WARP_SIZE-1);
        while (windex + offset <= maxR) {
            uint lixw = fromXform * d_g_Flame.numTrans + windex + offset;
            if (windex + offset > 0 && w >= d_g_switchMatrix[lixw - 1] && w < d_g_switchMatrix[lixw]) {
                rw[whichWarp] = windex + offset;
            }
            offset += WARP_SIZE;
        }
        __syncthreads();
        uint r = rw[whichWarp];
        toXform = r;
#else
        float w;
        w=randFloatWarp(randStates, whichWarp);
        uint r    = 0;
        while ((r < maxR) && (w > d_g_switchMatrix[fromXform * d_g_Flame.numTrans + r]))
        {
            r++;
        }
        toXform = r;
#endif
        //Now each thread chooses a point at random from the point pool.  This is done to allow each point to have a seperate xform path while retaining SIMD
        uint p               = shuffle[index + NUM_POINTS * j];
        uint fromPointIndex  = fromXform * xformPointPoolSize + NUM_POINTS*blockIndex + p;
        uint toPointIndex    = toXform   * xformPointPoolSize + NUM_POINTS*blockIndex + p;
        uint iterations      = pointIterations[fromPointIndex];
        uint varUsagesIndex  = d_g_varUsageIndexes[r];

        struct point fromPoint = points[fromPointIndex];

        //Iterate the chosen point and store it back to the pool
        iteratePoint(&d_g_varUsages[varUsagesIndex],
                     varpars,
                     &d_g_Xforms[r],
                     r,
                     epsilon,
                     &fromPoint,
                     activePoint,
                     randStates,
                     permutations,
#ifdef JWF_EXTENSIONS
                     palette,
                     numColors,
#endif
                     gradients);

#ifndef FOR_2D
        if (! isfinite(activePoint[index].x + activePoint[index].y + activePoint[index].z)) {
            // test to add back a random point (ala Flam3) to get Flam3 like images in borderline cases
            activePoint[index].x = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].y = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].z = 0; //2.f*randFloat(randStates) - 1.f;
            iterations           = 0;
        }

#else
        if (! isfinite(activePoint[index].x + activePoint[index].y)) {
            // test to add back a random point (ala Flam3) to get Flam3 like images in borderline cases
            activePoint[index].x = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].y = 2.f*randFloat(randStates) - 1.f;
            iterations           = 0;
        }

#endif
        ++iterations;
        struct point toPoint = activePoint[index]; // capture point state before final xform application
	

        if (iterations >= fuseIterations) { // dont store until fuse for each point is finished
            markedCounts[ix]++;  // keep track of number of iterations that could mark (versus unmarked because of unfused points)

            //Prepare the point for displey.  First the final transformation is applied
            if (d_g_Flame.useFinal) {
                for (uint fIndex = 0; fIndex < d_g_Flame.numFinal; fIndex++) {
                    uint varUsagesIndex  = d_g_varUsageIndexes[d_g_Flame.numTrans + fIndex];
                    struct point tempPoint = activePoint[index];
					float pal_save = activePoint[index].pal;
                    iteratePoint(&d_g_varUsages[varUsagesIndex],
                                 varpars,
                                 &d_g_Xforms[d_g_Flame.numTrans + fIndex],
                                 d_g_Flame.numTrans  + fIndex,
                                 epsilon,
                                 &tempPoint,
                                 activePoint,
                                 randStates,
                                 permutations,
#ifdef JWF_EXTENSIONS
                                 palette,
                                 numColors,
#endif
                                 gradients);
								activePoint[index].pal = pal_save; 
                }
            }

#ifdef JWF_EXTENSIONS
         if(!activePoint[index].doHide) {
#endif			

#ifndef FOR_2D
            projectJWF(&activePoint[index], d_g_Camera, RANDFLOAT(), RANDFLOAT());
            applyRotation(&activePoint[index], d_g_Camera->rotatedViewOffsetx, d_g_Camera->rotatedViewOffsety);
#else
            applyRotation( &activePoint[index], cosRotation, sinRotation);
#endif

            //Finally, we randomly jitter the point within a 1/2 pixel radius to obtain antialiasing
            float dr;
            dr = randFloat(randStates);
            dr = expf(d_g_Flame.supersampleWidth*sqrtf(-logf(dr)))-1.0f;
            float rn;
            rn = randFloat(randStates);
            float dtheta = (rn)*2.0f*PI;

            // mark the histogram
            float z = (d_g_Flame.clipToNDC != 0) ? activePoint[index].z : 0.f;
            int x,y;
            float sina, cosa;
            sincosf(dtheta, &sina, &cosa);

            x = floorf((((activePoint[index].x-d_g_Flame.center[0])/d_g_Flame.size[0]+.5f)*(float)xDim)+dr*cosa);
            y = floorf(((-(activePoint[index].y-d_g_Flame.center[1])/d_g_Flame.size[1]+.5f)*(float)yDim)+dr*sina);
            //And render the point to the accumulation buffer
            if ((z >= -1.f) && (z <= 1.f) && (x < xDim)&&(y < yDim)&&(x>=0)&&(y>=0))
            {
                float4 output;
#ifdef JWF_EXTENSIONS
             if(activePoint[index].useRgb) {
               output.x = activePoint[index].colorR;			 
               output.y = activePoint[index].colorG;			 
               output.z = activePoint[index].colorB;			 
               output.w = activePoint[index].colorA;			 
             }
             else {			 
#endif				
                if (paletteStepMode)
                    output = read_imageStepMode(palette, numColors, activePoint[index].pal);
                else
                    output = read_image(palette, numColors, activePoint[index].pal);
                // output = tex1D(texRef,activePoint[threadIdx.x].pal);
				
#ifdef JWF_EXTENSIONS
            } // if(activePoint[index].useRgb) { 
#endif				

#ifdef USE_ATOMICS
                float *ptr = (float *)&(renderTarget[y*xDim+x]);
                atomicAdd(ptr,     output.x*d_g_Xforms[r].opacity);
                atomicAdd(ptr + 1, output.y*d_g_Xforms[r].opacity);
                atomicAdd(ptr + 2, output.z*d_g_Xforms[r].opacity);
                atomicAdd(ptr + 3, output.w*d_g_Xforms[r].opacity);
                atomicAdd(&pixelCounts[y*xDim+x], 1);
#else
                renderTarget[y*xDim+x].x += output.x*d_g_Xforms[r].opacity;
                renderTarget[y*xDim+x].y += output.y*d_g_Xforms[r].opacity;
                renderTarget[y*xDim+x].z += output.z*d_g_Xforms[r].opacity;
                renderTarget[y*xDim+x].w += output.w*d_g_Xforms[r].opacity;
                pixelCounts[y*xDim+x]++;
#endif
            }
#ifdef JWF_EXTENSIONS
         } // if(activePoint[index].doHide==0) {
#endif			
			
        }
        pointIterations[toPointIndex] = iterations;
        points[toPointIndex]          = toPoint;
        fromXform                     = toXform;
        // essential to prevent cross-warp corruption with linear
    	__syncthreads();
        
    }
    startingXform[warpIndex] = toXform;
    perThreadRandSeeds[ix]   = randStates[index&(WARP_SIZE-1)];
}

extern "C" __global__ void postProcessStep1Kernal(
                                float4* renderTarget,
                                float4* accumBuffer,
                                uint xDim,
                                uint yDim,
                                int blocksY,
                                float fuseCompensation)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float k1 = (d_g_Flame.brightness*268.0f)/255.0f;
        float area = fabsf(d_g_Flame.size[0]*d_g_Flame.size[1]);
        float k2 = ((float)(xDim*yDim))/(area*fuseCompensation*((float)(NUM_ITERATIONS))*d_g_Flame.numBatches*32.f*1024.0f*((float)blocksY/32.f));
        float4 rgba = accumBuffer[iy*xDim+ix];
        float a = (k1* logf(1.0f+k2*rgba.w));
        float ls = a/rgba.w;
        rgba.x = ls*rgba.x;
        rgba.y = ls*rgba.y;
        rgba.z = ls*rgba.z;

        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void postProcessStep2Kernal(
                                float4* renderTarget,
                                float4* accumBuffer,
                                uint xDim,
                                uint yDim,
                                int blocksY,
                                float fuseCompensation,
                                float4 adjust)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float k1   = (d_g_Flame.brightness*268.0f)/255.0f;
        float area = fabsf(d_g_Flame.size[0]*d_g_Flame.size[1]);
        float _k2  = ((float)(xDim*yDim))/
                (area*fuseCompensation*((float)(NUM_ITERATIONS))*d_g_Flame.numBatches*32.f*1024.0f*((float)blocksY/32.f));
        float gammaThreshold = d_g_Flame.gammaThreshold;
        float gamma          = d_g_Flame.gamma;
        float alphaGamma     = d_g_Flame.alphaGamma;

        float4 k2   = make_float4(_k2/adjust.x, _k2/adjust.y, _k2/adjust.z, _k2/adjust.w);
        float4 rgba = accumBuffer[iy*xDim+ix];
        
        float4 a = make_float4(k1 * logf(1.0f + k2.x*rgba.w), 
                               k1 * logf(1.0f + k2.y*rgba.w), 
                               k1 * logf(1.0f + k2.z*rgba.w), 
                               k1 * logf(1.0f + k2.w*rgba.w));
                    
                                
        float4 fraction = make_float4(a.x/gammaThreshold, 
                                      a.y/gammaThreshold, 
                                      a.z/gammaThreshold, 
                                      a.w/gammaThreshold);
                                    
        float4 alpha = make_float4(powf(a.x, 1.0f/gamma-1.0f), 
                                    powf(a.y, 1.0f/gamma-1.0f), 
                                    powf(a.z, 1.0f/gamma-1.0f), 
                                    powf(a.w, 1.0f/gamma-1.0f));
                                    
        
        float alphaTx =  (1.f - fraction.x) * a.x * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.x * alpha.x;
        float alphaTy =  (1.f - fraction.y) * a.y * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.y * alpha.y;
        float alphaTz =  (1.f - fraction.z) * a.z * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.z * alpha.z;
        float alphaTw =  (1.f - fraction.w) * a.w * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.w * alpha.w;
                                    
        float4 alphaT   = make_float4(alphaTx, alphaTy, alphaTz, alphaTw);
        
        float4 ls = make_float4(
            d_g_Flame.vibrancy * (a.x < gammaThreshold ? alphaT.x : alpha.x),
            d_g_Flame.vibrancy * (a.y < gammaThreshold ? alphaT.y : alpha.y),
            d_g_Flame.vibrancy * (a.z < gammaThreshold ? alphaT.z : alpha.z),
            d_g_Flame.vibrancy * (a.w < gammaThreshold ? alphaT.w : alpha.w));

        float4 sign = make_float4(
            rgba.x >= 0.f ? 1.f : -1.f,
            rgba.y >= 0.f ? 1.f : -1.f,
            rgba.z >= 0.f ? 1.f : -1.f,
            rgba.w >= 0.f ? 1.f : -1.f);
       
        rgba.x        = ls.x*rgba.x + (1.0f-d_g_Flame.vibrancy)*sign.x*powf(fabsf(rgba.x), 1.0f/gamma);
        rgba.y        = ls.y*rgba.y + (1.0f-d_g_Flame.vibrancy)*sign.y*powf(fabsf(rgba.y), 1.0f/gamma);
        rgba.z        = ls.z*rgba.z + (1.0f-d_g_Flame.vibrancy)*sign.z*powf(fabsf(rgba.z), 1.0f/gamma);
        rgba.w        = ls.w*rgba.w + (1.0f-d_g_Flame.vibrancy)*sign.w*powf(fabsf(rgba.w), 1.0f/gamma);
        
        alpha.x       = powf(a.x, 1.0f/gamma);
        alpha.y       = powf(a.y, 1.0f/gamma);
        alpha.z       = powf(a.z, 1.0f/gamma);
        alpha.w       = powf(a.w, 1.0f/gamma);
        
        alphaT.x      = (1.f - fraction.x) * a.x * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.x * alpha.x;
        alphaT.y      = (1.f - fraction.y) * a.y * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.y * alpha.y;
        alphaT.z      = (1.f - fraction.z) * a.z * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.z * alpha.z;
        alphaT.w      = (1.f - fraction.w) * a.w * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.w * alpha.w;
        
        alpha.x      = fminf((a.x < gammaThreshold ? alphaT.x : alpha.x), 1.f);
        alpha.y      = fminf((a.y < gammaThreshold ? alphaT.y : alpha.y), 1.f);
        alpha.z      = fminf((a.z < gammaThreshold ? alphaT.z : alpha.z), 1.f);
        alpha.w      = fminf((a.w < gammaThreshold ? alphaT.w : alpha.w), 1.f);


        float alphaCw  = powf(a.w, 1.0f/alphaGamma);
        float alphaTCw =  (1.f - fraction.w) * a.w * (powf(gammaThreshold, alphaGamma)/gammaThreshold) + fraction.w * alphaCw;
        alphaCw        = fminf((a.w < gammaThreshold ? alphaTCw : alphaCw), 1.f);


        if (d_g_Flame.highlightPower >= 0.f) {
            rgba = RGBtoHSVHueAdjusted(rgba);
            if (rgba.z > 1.0f)
            {
                //rgba.y /= rgba.z;
                rgba.z = 1.0f;
            }
            rgba = HSVtoRGB(rgba);
        }
        if (isfinite(rgba.x + rgba.y + rgba.z + rgba.w))
        {
            renderTarget[iy*xDim+ix].x=rgba.x+renderTarget[iy*xDim+ix].x*(1.0f-alpha.x);
            renderTarget[iy*xDim+ix].y=rgba.y+renderTarget[iy*xDim+ix].y*(1.0f-alpha.y);
            renderTarget[iy*xDim+ix].z=rgba.z+renderTarget[iy*xDim+ix].z*(1.0f-alpha.z);
            renderTarget[iy*xDim+ix].w=alphaCw;
        }
        else {
            renderTarget[iy*xDim+ix].w=0.f;
        }
    }
}

extern "C" __global__ void colorCurveRGB3ChannelsKernal(float4* accumBuffer,
                                        float* X,
                                        float* A,
                                        float* B,
                                        float* C,
                                        float* D,
                                        uint xDim,
                                        uint yDim,
                                        uint cpCount)

{
    __shared__ float xs[256];
    __shared__ float as[256];
    __shared__ float bs[256];
    __shared__ float cs[256];
    __shared__ float ds[256];

    cpCount    = cpCount > 256 ? 256 : cpCount;
    uint index = threadIdx.x;

    for (uint i = 0; index+i < cpCount; i += blockDim.x) {
        xs[index+i] = X[index+i];
        as[index+i] = A[index+i];
        cs[index+i] = C[index+i];
    }
    for (uint i = 0; index+i < cpCount-1; i += blockDim.x) {
        bs[index+i] = B[index+i];
        ds[index+i] = D[index+i];
    }
	__syncthreads();

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float4 rgba = accumBuffer[iy*xDim+ix];
        // sRGB luma   0.212656 R  0.715158 G  0.072186 B
        // AdobeRGB luma
        float preluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        preluma       /= rgba.w;
        float postluma = curveAdjust(preluma, xs, as, bs, cs, ds, cpCount);

        if (preluma != 0.f) {
            rgba.x = postluma/preluma * rgba.x;
            rgba.y = postluma/preluma * rgba.y;
            rgba.z = postluma/preluma * rgba.z;
            rgba.w = postluma/preluma * rgba.w;
        }
        else {
            rgba.x = postluma;
            rgba.y = postluma;
            rgba.z = postluma;
            rgba.w = postluma;
        }
        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void colorCurveRGBChannelKernal(float4* accumBuffer,
                                        float* X,
                                        float* A,
                                        float* B,
                                        float* C,
                                        float* D,
                                        uint xDim,
                                        uint yDim,
                                        uint cpCount,
                                        uint channel)
{
    __shared__ float xs[256];
    __shared__ float as[256];
    __shared__ float bs[256];
    __shared__ float cs[256];
    __shared__ float ds[256];

    cpCount    = cpCount > 256 ? 256 : cpCount;
    uint index = threadIdx.x;

    for (uint i = 0; index+i < cpCount; i += blockDim.x) {
        xs[index+i] = X[index+i];
        as[index+i] = A[index+i];
        cs[index+i] = C[index+i];
    }
    for (uint i = 0; index+i < cpCount-1; i += blockDim.x) {
        bs[index+i] = B[index+i];
        ds[index+i] = D[index+i];
    }
	__syncthreads();

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float4 rgba = accumBuffer[iy*xDim+ix];

        float preluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        switch (channel) {
            default:
            case 0:
            rgba.x = rgba.w * curveAdjust(rgba.x/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 1:
            rgba.y = rgba.w * curveAdjust(rgba.y/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 2:
            rgba.z = rgba.w * curveAdjust(rgba.z/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 3:
            break;
        }
        float postluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        // maintain same luminance afterwards
        if (preluma != 0)
            rgba.w *= postluma/preluma;
        else
            rgba.w  = postluma;

        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void setBufferKernal(float4* renderTarget, float4 value, uint xDim, uint yDim)
{
    //This kernal simply fills the render target with value
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;

    if ((ix < xDim)&&(iy < yDim))
        renderTarget[iy*xDim+ix] = value;
}

extern "C" __global__ void FlexibleDensityEstimationKernal(float* output, float* input, unsigned int xDim, unsigned int yDim,
                                                            float baseThreshold, int radius)
{
    __shared__ float filterLocal[4*(DENSITY_KERNAL_RADIUS*2+16)*(DENSITY_KERNAL_RADIUS*2+16)];
    unsigned int index    = threadIdx.x;
    const unsigned int ix = blockDim.x*blockIdx.x + index;
    const unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;
    const unsigned int lidx = (blockDim.x+radius*2)*threadIdx.y + index;
    //First, we load up the block of pixels we will be working on into shared memory
    for (unsigned int y = 0; y < blockDim.y+radius*2-threadIdx.y; y += blockDim.y)
    {
        for(unsigned int x = 0; x < blockDim.x+radius*2-index; x += blockDim.x)
        {
            unsigned toIndex4   = 4*(lidx+x+y*(blockDim.x+radius*2));
            //clamped addressing
            unsigned fromIndex4 = 4*(max(min(ix+x-radius,xDim-1),0U)+max(min(iy+y-radius,yDim-1),0U)*xDim);
            filterLocal[toIndex4]     = input[fromIndex4];
            filterLocal[toIndex4 + 1] = input[fromIndex4 + 1];
            filterLocal[toIndex4 + 2] = input[fromIndex4 + 2];
            filterLocal[toIndex4 + 3] = input[fromIndex4 + 3];
        }
    }
    __syncthreads();
    //Next, apply the actual filter
    if ((ix < xDim)&& (iy < yDim))
    {
        unsigned idx = 4*(ix+iy*xDim);
        float pntw = input[idx + 3];
        float sumx = 0.f;
        float sumy = 0.f;
        float sumz = 0.f;
        float sumw = 0.f;

        float count = 0.0f;
        for (int y = 0; y < radius*2+1; y++)
        {
            for (int x = 0; x<radius*2+1; x++)
            {
                int cellIdx   = 4*(lidx+x+y*(blockDim.x+2*radius));
                float invDist = 1.0f/(((float)(x-radius)*(float)(x-radius)+(float)(y-radius)*(float)(y-radius))+1.0f);
                float deviation = fabsf(erff((filterLocal[cellIdx + 3]-pntw)/(sqrtf(8.0f*pntw)+5.0f)));
                if (deviation<=powf(baseThreshold*.9f,sqrtf(1.0f/invDist))*powf(pntw+1.0f,-0.25f))
                {
                    sumx += filterLocal[cellIdx]*invDist;
                    sumy += filterLocal[cellIdx + 1]*invDist;
                    sumz += filterLocal[cellIdx + 2]*invDist;
                    sumw += filterLocal[cellIdx + 3]*invDist;
                    count += invDist;
                }
            }
        }
        sumx/=count;
        sumy/=count;
        sumz/=count;
        sumw/=count;
        //And store the result
        output[idx]     = isfinite(sumx)  ? sumx : 0.f;
        output[idx + 1] =  isfinite(sumy) ? sumy : 0.f;
        output[idx + 2] =  isfinite(sumz) ? sumz : 0.f;
        output[idx + 3] =  isfinite(sumw) ? sumw : 0.f;
    }
}

extern "C" __global__ void RGBA128FtoRGBA32UKernal(uchar4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    //This kernal converts a 32bit per channel floating point image to a 8bit per channel integer image
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_uchar4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*255.0f);
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*255.0f;
                output[iy*xDim+ix] = make_uchar4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    alpha);
            }
            else
                output[iy*xDim+ix]=make_uchar4(0,0,0,0);
            }
        else
        {
        if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
            output[iy*xDim+ix] = make_uchar4(
                fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*255.0f,
                fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*255.0f,
                fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*255.0f,
                255);
        else
            output[iy*xDim+ix]=make_uchar4(0,0,0,255);
        }
    }
}

extern "C" __global__ void RGBA128FtoBGRA32UKernal(uchar4* output, float4* input, uint xDim, uint yDim)
{
    //This kernal converts a 32bit per channel floating point image to a 8bit per channel integer image
    // in BGRA format for little endian Intel with premultiplied alpha
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
        output[iy*xDim+ix] = make_uchar4(
            fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*255.0f, // blue
            fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*255.0f, // green
            fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*255.0f, // red
            255);
        else
            output[iy*xDim+ix]=make_uchar4(0,0,0,255);
    }
}

extern "C" __global__ void RGBA128FtoRGBA64UKernal(ushort4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    //This kernal converts a 32bit per channel floating point image to a 16bit per channel integer image
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_ushort4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f);
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f;
                output[iy*xDim+ix] = make_ushort4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    alpha);
            }
            else
                output[iy*xDim+ix]=make_ushort4(0,0,0,0);
        }
        else
        {
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
                output[iy*xDim+ix] = make_ushort4(
                    fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*65535.0f,
                    fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*65535.0f,
                    fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*65535.0f,
                    65535);
            else
                output[iy*xDim+ix]=make_ushort4(0,0,0,65535);
        }
    }
}

extern "C" __global__ void RGBA128FtoRGBA128FKernal(float4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_float4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f));
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f);
                output[iy*xDim+ix] = make_float4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                alpha);
            }
            else
                output[iy*xDim+ix]=make_float4(0.f,0.f,0.f,0.f);
        }
        else
        {
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
                output[iy*xDim+ix] = make_float4(
                    fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f),
                    fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f),
                    fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f),
                    1.f);
            else
                output[iy*xDim+ix]=make_float4(0.f,0.f,0.f,1.f);
        }
    }
}

extern "C" __global__ void MergeKernal(float4* accum, float4* input, uint xDim, uint yDim)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        accum[iy*xDim+ix].x += input[iy*xDim+ix].x;
        accum[iy*xDim+ix].y += input[iy*xDim+ix].y;
        accum[iy*xDim+ix].z += input[iy*xDim+ix].z;
        accum[iy*xDim+ix].w += input[iy*xDim+ix].w;
    }
}

extern "C" __global__ void readChannelKernel(float* output, float4* input, uint xDim, uint yDim, uint channel)
{
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        switch (channel) {
            default:
            case 0:
            output[iy*xDim+ix] = input[iy*xDim+ix].x;
            break;
            case 1:
            output[iy*xDim+ix] = input[iy*xDim+ix].y;
            break;
            case 2:
            output[iy*xDim+ix] = input[iy*xDim+ix].z;
            break;
            case 3:
            output[iy*xDim+ix] = input[iy*xDim+ix].w;
            break;
        }
    }
}

extern "C" __global__ void writeChannelKernel(float4* output, float* input, uint xDim, uint yDim, uint channel)
{
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        switch (channel) {
            default:
            case 0:
            output[iy*xDim+ix].x = input[iy*xDim+ix];
            break;
            case 1:
            output[iy*xDim+ix].y = input[iy*xDim+ix];
            break;
            case 2:
            output[iy*xDim+ix].z = input[iy*xDim+ix];
            break;
            case 3:
            output[iy*xDim+ix].w = input[iy*xDim+ix];
            break;
        }
    }
}

extern "C" __global__ void writeChannelStripedKernel(float4* output,
                                        float* input,
                                        uint xDim,
                                        uint yDim,
                                        uint channel,
                                        uint supersample)
{
    const uint resampledXdim = xDim / supersample;
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim) && (iy < yDim) && (ix % supersample == 0) && (iy % supersample == 0))
    {
        const uint x = ix / supersample;
        const uint y = iy / supersample;
        switch (channel) {
            default:
            case 0:
            output[y*resampledXdim+x].x = input[iy*xDim+ix];
            break;
            case 1:
            output[y*resampledXdim+x].y = input[iy*xDim+ix];
            break;
            case 2:
            output[y*resampledXdim+x].z = input[iy*xDim+ix];
            break;
            case 3:
            output[y*resampledXdim+x].w = input[iy*xDim+ix];
            break;
        }
    }
}


///////////////////////////////////////////////////////////////////////////////
/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
* OpenCL port & resampling kernels Copyright 2014 Centcom Inc. All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.  Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.   This source code is a "commercial item" as
* that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer  software"  and "commercial computer software
* documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*
* Any use of this source code in individual and commercial software must
* include, in the user documentation and internal comments to the code,
* the above Disclaimer and U.S. Government End Users Notice.
*/

////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////

// Assuming ROW_TILE_W, KERNEL_RADIUS_ALIGNED and dataW
// are multiples of coalescing granularity size,
// all global memory operations are coalesced in convolutionRowGPU()
#define            ROW_TILE_W 128
#define KERNEL_RADIUS_ALIGNED 16

// Assuming COLUMN_TILE_W and dataW are multiples
// of coalescing granularity size, all global memory operations
// are coalesced in convolutionColumnGPU()
#define COLUMN_TILE_W 16
#define COLUMN_TILE_H 48

extern "C" __global__ void convolutionRowsKernel(float *d_Kernel,
                                float *d_Result,
                                float *d_Data,
                                int dataW,
                                int dataH,
                                int KERNEL_RADIUS
){
    //Data cache
    __shared__ float data[KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS_ALIGNED];

    //Current tile and apron limits, relative to row start
    const int         tileStart = blockIdx.x * ROW_TILE_W;
    const int           tileEnd = tileStart + ROW_TILE_W - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataW - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataW - 1);

    //Row start index in d_Data[]
    const int          rowStart = blockIdx.y * dataW;

    //Aligned apron start. Assuming dataW and ROW_TILE_W are multiples
    //of half-warp size, rowStart + apronStartAligned is also a
    //multiple of half-warp size, thus having proper alignment
    //for coalesced d_Data[] read.
    const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;

    const int loadPos = apronStartAligned + threadIdx.x;
    //Set the entire data cache contents
    //Load global memory values, if indices are within the image borders,
    //or initialize with zeroes otherwise
    if(loadPos >= apronStart){
        const int smemPos = loadPos - apronStart;

        // out of bounds set to 0
        // data[smemPos] =
        //     ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ? d_Data[rowStart + loadPos] : 0;

        // reflected at boundary
        // data[smemPos] =
        // loadPos < apronStartClamped ? d_Data[rowStart + apronStartClamped + apronStartClamped - loadPos] :
        //  (loadPos > apronEndClamped ? d_Data[rowStart + apronEndClamped   + apronEndClamped   - loadPos] :
        //     d_Data[rowStart + loadPos]);

        // clamp to border
        data[smemPos] =
            loadPos < apronStartClamped ? d_Data[rowStart + apronStartClamped] :
            (loadPos > apronEndClamped ? d_Data[rowStart + apronEndClamped] :
            d_Data[rowStart + loadPos]);
    }


    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
	__syncthreads();

    const int writePos = tileStart + threadIdx.x;

    //Assuming dataW and ROW_TILE_W are multiples of half-warp size,
    //rowStart + tileStart is also a multiple of half-warp size,
    //thus having proper alignment for coalesced d_Result[] write.
    if(writePos <= tileEndClamped){
        const int smemPos = writePos - apronStart;
        float sum = 0;

        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += data[smemPos + k] * d_Kernel[KERNEL_RADIUS - k];

        d_Result[rowStart + writePos] = isfinite(sum) ? sum : 0.f;
    }
}

extern "C" __global__ void convolutionColumnsKernel(float *d_Kernel,
                                        float *d_Result,
                                        float *d_Data,
                                        int dataW,
                                        int dataH,
                                        int smemStride,
                                        int gmemStride,
                                        int KERNEL_RADIUS
){
    //Data cache
    __shared__ float data[COLUMN_TILE_W * (KERNEL_RADIUS_ALIGNED + COLUMN_TILE_H + KERNEL_RADIUS_ALIGNED)];

    //Current tile and apron limits, in rows
    const int         tileStart = blockIdx.y * COLUMN_TILE_H;
    const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataH - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataH - 1);

    //Current column index
    const int       columnStart = blockIdx.x * COLUMN_TILE_W + threadIdx.x;

    //Shared and global memory indices for current column
    int smemPos    = threadIdx.y * COLUMN_TILE_W + threadIdx.x;
    int gmemPos    = (apronStart + threadIdx.y) * dataW + columnStart;
    int gmemPosMin = columnStart;
    int gmemPosMax = columnStart + dataW * (dataH - 1);

    //Cycle through the entire data cache
    //Load global memory values, if indices are within the image borders,
    //or initialize with zero otherwise
    for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
        // out of bounds set to 0
        //data[smemPos] =
        //    ((y >= apronStartClamped) && (y <= apronEndClamped)) ? d_Data[gmemPos] : 0;

        // reflected at boundary
        // data[smemPos] =
        //     y < apronStartClamped ? d_Data[gmemPosMin + (apronStartClamped - y) * dataW] :
        //    (y > apronEndClamped   ? d_Data[gmemPosMax + (apronEndClamped   - y) * dataW] : d_Data[gmemPos]);

        // clamp to border
        data[smemPos] =
            y < apronStartClamped ? d_Data[gmemPosMin] :
            (y > apronEndClamped   ? d_Data[gmemPosMax] : d_Data[gmemPos]);

        smemPos += smemStride;
        gmemPos += gmemStride;
    }

    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
	__syncthreads();

    //Shared and global memory indices for current column
    smemPos = (threadIdx.y + KERNEL_RADIUS) * COLUMN_TILE_W + threadIdx.x;
    gmemPos = (tileStart + threadIdx.y) * dataW + columnStart;

    //Cycle through the tile body, clamped by image borders
    //Calculate and output the results
    for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
        float sum = 0;

        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += data[smemPos + k * COLUMN_TILE_W] * d_Kernel[KERNEL_RADIUS - k];

        d_Result[gmemPos] = isfinite(sum) ? sum : 0.f;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }
}
